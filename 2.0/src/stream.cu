#include "codec.cu"

/**
 * Chunk represents a unit of data read by the stream
 */
template <typename T> class Chunk {
public:
	T* ptr = NULL;
	size_t len = 0;
	bool not_null() {return ptr != NULL;}
};

/**
 * A stream where the data is already fit inside the GPU memory, so
 * streaming is basically just shift the pointer around.
 */
template <typename T> class GPUInputStream {
private:
	T* _data = NULL;
	size_t _len = 0, _chunkSize = 0, _tp;

	void check_input(T* data, size_t len, size_t chunkSize) {
		if (data == NULL)
			print_err("GPUInputStream: data == NULL");
		if (len <= 0)
			print_err("GPUInputStream: len <= 0");
		if (chunkSize <= 0)
			print_err("GPUInputStream: chunkSize <= 0");
	}

public:
	GPUInputStream(T* data, size_t len, size_t chunkSize) {
		check_input(data, len, chunkSize);
		_data = data;
		_len = len;
		_chunkSize = chunkSize;
		_tp = len;
	}

	Chunk<T> read() {
		Chunk<T> ans;
		if (_len == 0)
			return ans;

		ans.ptr = _data;
		ans.len = _chunkSize > _len ? _len : _chunkSize;
		_len -= ans.len;
		_data += ans.len;
		return ans;
	}

	size_t get_throughput() {
		return _tp;
	}
};

/**
 * A 1 dimensional stream where the original data resides in the RAM
 * and a chunk is transfered to GPU memory one at a time.
 *
 * Note that the RAM backend is designed to be shared with RAMOutputStream.
 */
template <typename T> class RAMInputStream {
private:
	T** _data = NULL;
	size_t* _len2 = NULL;
	size_t _len1 = 0, _maxReadableSize = 0, _index = 0;
	T* _deviceBuffer = NULL;

	void check_input(T** data, size_t len1, size_t* len2, size_t maxReadableSize, T* deviceBuffer) {
		if (len1 <= 0)
			print_err("RAMInputStream: len1 <= 0");
		if (maxReadableSize <= 0)
			print_err("RAMInputStream: maxReadableSize <= 0");
		if (data == NULL)
			print_err("RAMInputStream: data == NULL");
		if (deviceBuffer == NULL)
			print_err("RAMInputStream: deviceBuffer == NULL");
		for (size_t i = 0; i < len1; i++) {
			if (len2[i] > maxReadableSize)
				print_err("RAMInputStream: len2[i] > maxReadableSize will lead to infinite loop");
			if ((data[i] == NULL) && (len2[i] > 0))
				print_err("RAMInputStream: (data[i] == NULL) && (len2[i] > 0)");
		}
	}

public:
	RAMInputStream(T** data, size_t len1, size_t* len2, size_t maxReadableSize, T* deviceBuffer) {
		check_input(data, len1, len2, maxReadableSize, deviceBuffer);
		_data = data;
		_len1 = len1;
		_len2 = len2;
		_maxReadableSize = maxReadableSize;
		_deviceBuffer = deviceBuffer;
	}

	Chunk<T> read() {
		Chunk<T> ans;
		if (_index == _len1)
			return ans;

		ans.ptr = _deviceBuffer;
		T* currentPtr = _deviceBuffer;
		for (; _index < _len1; _index++) {
			size_t newLen = _len2[_index];
			if (ans.len + newLen > _maxReadableSize)
				break;
			if (newLen == 0)
				continue;

			hipMemcpy(currentPtr, _data[_index], sizeof(T)*newLen, hipMemcpyHostToDevice);
			currentPtr += newLen;
			ans.len += newLen;
		}
		return ans;
	}

	size_t get_throughput() {
		size_t ans = 0;
		for (size_t i = 0; i < _len1; i++)
			ans += _len2[i];
		return ans;
	}
};

/**
 * A 1 dimensional stream where the original data resides in the RAM
 * and a chunk is written from GPU memory one at a time.
 *
 * Note that the RAM backend is designed to be shared with RAMInputStream.
 */
template <typename T> class RAMOutputStream {
private:
	T** _data = NULL;
	size_t* _len2 = NULL;
	size_t _len1 = 0, _index = 0;

	void check_input(T** data, size_t len1, size_t* len2) {
		if (len1 <= 0)
			print_err("RAMOutputStream: len1 <= 0");
		if (data == NULL)
			print_err("RAMOutputStream: data == NULL");
	}

	void check_input_write(T* newData, size_t n) {
		if ((newData == NULL) && (n > 0))
			print_err("RAMOutputStream: (newData == NULL) && (n > 0)");
	}

public:
	RAMOutputStream(T** data, size_t len1, size_t* len2) {
		check_input(data, len1, len2);
		_data = data;
		_len1 = len1;
		_len2 = len2;
	}

	void write(T* newData, size_t n) {
		check_input_write(newData, n);
		if (_index >= _len1)
			print_err("RAMOutputStream: writing more than allocated");
		if (n > 0)
			hipMemcpy(_data[_index], newData, sizeof(T)*n, hipMemcpyDeviceToHost);
		_len2[_index] = n;
		_index++;
	}

	size_t get_new_len1() {
		return _index;
	}

	size_t* get_new_len2() {
		return _len2;
	}

	size_t get_throughput() {
		size_t ans = 0;
		for (size_t i = 0; i < _len1; i++)
			ans += _len2[i];
		return ans;
	}
};

/**
 * A 2 dimensional stream where the data is written in row direction and read in column direction.
 */
template <typename T> class D2Stream {
private:
	T** _data;
	size_t _len1, _offset_len;
	size_t _write_index = 0, _read_index = 0;
	size_t* _len2;
	size_t** _offsets = NULL;
	T* _deviceBuffer = NULL;

	void check_input(size_t len1) {
		if (len1 <= 0)
			print_err("D2Stream: len1 <= 0");
	}

	void check_input_write(T* newData, size_t n) {
		if ((n != 0) && (newData == NULL))
			print_err("D2Stream: (n != 0) && (newData == NULL)");
	}

	void check_input_offsets(size_t** offsets, size_t offset_len) {
		if (offsets == NULL)
			print_err("D2Stream: offsets == NULL");
		for (int i = 0; i < _len1; i++) {
			if (offsets[i] == NULL)
				print_err("D2Stream: offsets[i]==NULL");
			if (offsets[i][offset_len - 1] != _len2[i])
				print_err("D2Stream: the last offset should cover the whole stream");
		}
	}

public:
	D2Stream(size_t len1) {
		check_input(len1);
		_len1 = len1;
		hipHostMalloc((void**)&_data, sizeof(T*)*len1);
		hipHostMalloc((void**)&_len2, sizeof(size_t)*len1);
	}

	void write(T* newData, size_t n) {
		check_input_write(newData, n);
		if (_write_index > _len1)
			print_err("D2Stream: writing more than allocated");
		if (n > 0) {
			hipHostMalloc((void**) &_data[_write_index], sizeof(T)*n);
			hipMemcpy(_data[_write_index], newData, sizeof(T)*n, hipMemcpyDeviceToHost);
		}
		_len2[_write_index] = n;
		_write_index++;
	}

	void set_offsets(size_t** offsets, size_t offset_len) {
		check_input_offsets(offsets, offset_len);
		if (_deviceBuffer != NULL)
			print_err("D2Stream: set_offsets is called more than once");

		_offsets = offsets;
		_offset_len = offset_len;

		// find minimum size of deviceBuffer and allocate it
		size_t maxLength = 0;
		for (size_t i = 0; i < offset_len; i++) {
			size_t newLength = 0;
			for (size_t j = 0; j < _len1; j++) {
				size_t start = i == 0 ? 0 : offsets[j][i - 1];
				newLength += offsets[j][i] - start;
			}
			if (newLength > maxLength)
				maxLength = newLength;
		}
		hipMalloc((void**)&_deviceBuffer, sizeof(T)*maxLength);
	}

	Chunk<T> read() {
		Chunk<T> ans;
		if (_write_index != _len1)
			print_err("D2Stream: read is called before fully written");
		if (_offsets == NULL)
			print_err("D2Stream: _offsets == NULL");
		if (_read_index == _offset_len)
			return ans;

		ans.ptr = _deviceBuffer;
		T * currentPtr = _deviceBuffer;
		for (size_t i = 0; i < _len1; i++) {
			size_t start = _read_index == 0 ? 0 : _offsets[i][_read_index - 1];
			size_t chunkLen = _offsets[i][_read_index] - start;
			if (chunkLen <= 0)
				continue;

			hipMemcpy(currentPtr, _data[i] + start, sizeof(T)*chunkLen, hipMemcpyHostToDevice);
			currentPtr += chunkLen;
			ans.len += chunkLen;
		}

		_read_index++;
		return ans;
	}

	void deconstruct() {
		for (int i = 0; i < _write_index; i++)
			hipHostFree(_data[i]);
		_cudaFreeHost(_data, _len2);
		hipFree(_deviceBuffer);
		_offsets = NULL;
	}

	size_t get_throughput() {
		size_t ans = 0;
		for (size_t i = 0; i < _len1; i++)
			ans += _len2[i];
		return ans;
	}
};
