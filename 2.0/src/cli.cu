#include "hip/hip_runtime.h"
#include <stdio.h>
#include <locale.h>
#include "xtn.cu"

FILE* outputFile = NULL; /*global variable for callback*/
size_t totalOutputLen = 0; /*global variable for callback*/
const char VERSION[] = "2.0.0\n";
const char HELP_TEXT[] = "xt_neighbor: perform either nearest neighbor search for CDR3 sequences or immune repertoire overlap using GPU-based xt_neighbor algorithm.\n"
                         "\t====================\n\t Common Options\n\t====================\n"
                         "\t -d or --distance [number]: distance threshold defining the neighbor (default to 1)\n"
                         "\t -o or --output-path [str]: path of the output file (default to no output)\n"
                         "\t -m or --measurement [leven|hamming]: distance measurement (default to leven)\n"
                         "\t -v or --version: print the version of the program then exit\n"
                         "\t -h or --help: print the help text of the program then exit\n"
                         "\t -V or --verbose: print extra detail as the program runs for debugging purpose\n"
                         "\t====================\n\t Nearest Neighbor Options\n\t====================\n"
                         "\t -i or --input-path [str] (required): path of csv input file containing a single column of CDR3 amino acid sequences\n"
                         "\t -n or --input-length [number] (required): number of rows given in the input file\n"
                         "\t====================\n\t Repertoire Overlap Options\n\t====================\n"
                         "\t -i or --input-path [str] (required): path of csv input file containing 2 columns: CDR3 amino acid sequences and their frequency. Note that the sequences are assumed to be unique\n"
                         "\t -n or --input-length [number] (required): number of sequences given in the input file\n"
                         "\t -r or --repertoire-info [str] (required): path of csv input file containing 2 columns: repertoire names, and their sizes. Note that the order of input sequence must be sorted according to this repertoire info\n"
                         "\t -N or --info-length [number] (required): number of repertoires given in the input file\n"
                         ;

int parse_args(int argc, char **argv, XTNArgs* ans) {
	char* current;

	for (int i = 1; i < argc; i++) {
		current = argv[i];

		if (strcmp(current, "-v") == 0 || strcmp(current, "--version") == 0) {
			printf("%s", VERSION);
			return EXIT;
		}
		else if (strcmp(current, "-h") == 0 || strcmp(current, "--help") == 0) {
			printf("%s", HELP_TEXT);
			return EXIT;
		}
		else if (strcmp(current, "-V") == 0 || strcmp(current, "--verbose") == 0)
			ans->verbose = 1;
		else if (strcmp(current, "-p") == 0 || strcmp(current, "--input-path") == 0)
			ans->seq1Path = argv[++i];
		else if (strcmp(current, "-o") == 0 || strcmp(current, "--output-path") == 0)
			ans->outputPath = argv[++i];
		else if (strcmp(current, "-d") == 0 || strcmp(current, "--distance") == 0) {
			int distance = ans->distance = atoi(argv[++i]);
			if (distance < 1 || distance > MAX_DISTANCE)
				return print_err("distance must be a valid number ranging from 1-2");
		}
		else if (strcmp(current, "-n") == 0 || strcmp(current, "--input-length") == 0) {
			ans->seq1Len = atoi(argv[++i]);
			if (ans->seq1Len == 0)
				return print_err("invalid input length");
		}
		else if (strcmp(current, "-m") == 0 || strcmp(current, "--measurement") == 0) {
			char* measure = argv[++i];
			if (strcmp(measure, "leven") == 0)
				ans->measure = LEVENSHTEIN;
			else if (strcmp(measure, "hamming") == 0)
				ans->measure = HAMMING;
			else
				return print_err("invalid measure option");
		}
		else
			return print_err("unknown option");
	}

	if (ans->seq1Path == NULL)
		return print_err("missing path for seq1");
	if (ans->seq1Len == 0)
		return print_err("missing length for seq1");

	return SUCCESS;
}

/**
 * read and parse text file to Int3*
*/
int parse_file(char* path, Int3* result, int len) {
	FILE* file = fopen(path, "r");
	if (file == NULL)
		return print_err("file reading failed");

	const int BUFFER_SIZE = 50;
	char line[BUFFER_SIZE];
	int lineNumber = 0, inputCount = 0;
	Int3 newInt3;

	while (fgets(line, BUFFER_SIZE, file)) {
		lineNumber++;
		if (strcmp(line, "\n") == 0 || strcmp(line, " \n") == 0)
			continue;

		newInt3 = str_encode(line);
		if (newInt3.entry[0] == 0) {
			fclose(file);
			char msg[100];
			sprintf(msg, "parsing error at line: %d (only upper-cased amino acids with max length of %d are allowed)", lineNumber, MAX_INPUT_LENGTH);
			return print_err(msg);
		}

		result[inputCount++] = newInt3;
	}

	if (inputCount != len)
		return print_err("input length doesn't match with the actual");

	fclose(file);
	return SUCCESS;
}

void null_handler(XTNOutput output) {
	totalOutputLen += output.len;
}

void file_handler(XTNOutput output) {
	Int2 current;
	for (int i = 0; i < output.len; i++) {
		current = output.indexPairs[i];
		fprintf(outputFile, "%d %d %d\n", current.x, current.y , output.pairwiseDistances[i]);
	}
	totalOutputLen += output.len;
}

int main(int argc, char **argv) {
	int returnCode;
	XTNArgs args;
	Int3* seq1;

	// 1. parse command line arguments
	setlocale(LC_ALL, "");
	returnCode = parse_args(argc, argv, &args);
	verboseGlobal = args.verbose;
	if (returnCode != SUCCESS)
		return returnCode;

	// 2. read input
	hipHostMalloc(&seq1, sizeof(Int3) * args.seq1Len); gpuerr();
	returnCode = parse_file(args.seq1Path, seq1, args.seq1Len);
	if (returnCode != SUCCESS) {
		hipFree(seq1); gpuerr();
		return returnCode;
	}
	if (verboseGlobal)
		print_args(args);

	// 3. perform algorithm
	if (args.outputPath != NULL) {
		if (outputFile != NULL)
			return print_err("output file has already been allocated, possibly due to concurrency");
		outputFile = fopen(args.outputPath, "w");
		if (outputFile == NULL)
			return print_err("file reading failed");
		xtn_perform(args, seq1, file_handler);
		fclose(outputFile);
	} else {
		xtn_perform(args, seq1, null_handler);
	}
	printf("total output length: %'lu\n", totalOutputLen);

	// 4. clean up
	hipHostFree(seq1); gpuerr();
	return 0;
}