#include "hip/hip_runtime.h"
#include <stdio.h>
#include <locale.h>
#include "xtn.cu"

FILE* outputFile = NULL; /*global variable for callback*/
size_t totalOutputLen = 0; /*global variable for callback*/

const char VERSION[] = "2.0.0\n";
const char HELP_TEXT[] = "xt_neighbor: perform either nearest neighbor search for CDR3 sequences or immune repertoire overlap using GPU-based xt_neighbor algorithm.\n"
                         "\t====================\n\t Common Options\n\t====================\n"
                         "\t -d or --distance [number]: distance threshold defining the neighbor (default to 1)\n"
                         "\t -o or --output-path [str]: path of the output file (default to no output)\n"
                         "\t -m or --measurement [leven|hamming]: distance measurement (default to leven)\n"
                         "\t -v or --version: print the version of the program then exit\n"
                         "\t -h or --help: print the help text of the program then exit\n"
                         "\t -V or --verbose: print extra detail as the program runs for debugging purpose\n"
                         "\t -x or --extended-buffer: switch to the alternate buffer implementation that can be enlarged using disc memory\n"
                         "\t====================\n\t Nearest Neighbor Options\n\t====================\n"
                         "\t -i or --input-path [str] (required): path of csv input file containing exactly 1 column: CDR3 amino acid sequences\n"
                         "\t -n or --input-length [number] (required): number of rows given in the input file\n"
                         "\t====================\n\t Repertoire Overlap Options\n\t====================\n"
                         "\t -i or --input-path [str] (required): path of csv input file containing exactly 2 columns: CDR3 amino acid sequences and their frequency. Note that the sequences are assumed to be unique\n"
                         "\t -n or --input-length [number] (required): number of sequences given in the input file\n"
                         "\t -r or --info-path [str] (required): path of csv input file containing exactly 1 column: repertoire sizes. Note that the order of input sequence must be sorted according to this repertoire info\n"
                         "\t -N or --info-length [number] (required): number of repertoires given in the info file\n"
                         ;

int parse_args(int argc, char **argv, XTNArgs* ans) {
	char* current;

	for (int i = 1; i < argc; i++) {
		current = argv[i];

		if (strcmp(current, "-v") == 0 || strcmp(current, "--version") == 0) {
			printf("%s", VERSION);
			return EXIT;
		}
		else if (strcmp(current, "-h") == 0 || strcmp(current, "--help") == 0) {
			printf("%s", HELP_TEXT);
			return EXIT;
		}
		else if (strcmp(current, "-V") == 0 || strcmp(current, "--verbose") == 0)
			ans->verbose = 1;
		else if (strcmp(current, "-x") == 0 || strcmp(current, "--extended-buffer") == 0)
			ans->extendedBuffer = 1;
		else if (strcmp(current, "-p") == 0 || strcmp(current, "--input-path") == 0)
			ans->seq1Path = argv[++i];
		else if (strcmp(current, "-r") == 0 || strcmp(current, "--info-path") == 0)
			ans->infoPath = argv[++i];
		else if (strcmp(current, "-o") == 0 || strcmp(current, "--output-path") == 0)
			ans->outputPath = argv[++i];
		else if (strcmp(current, "-d") == 0 || strcmp(current, "--distance") == 0) {
			int distance = ans->distance = atoi(argv[++i]);
			if (distance < 1 || distance > MAX_DISTANCE)
				return print_err("distance must be a valid number ranging from 1-2");
		}
		else if (strcmp(current, "-n") == 0 || strcmp(current, "--input-length") == 0) {
			ans->seq1Len = atoi(argv[++i]);
			if (ans->seq1Len == 0)
				return print_err("invalid input length");
		}
		else if (strcmp(current, "-N") == 0 || strcmp(current, "--info-length") == 0) {
			ans->infoLen = atoi(argv[++i]);
			if (ans->infoLen == 0)
				return print_err("invalid info length");
		}
		else if (strcmp(current, "-m") == 0 || strcmp(current, "--measurement") == 0) {
			char* measure = argv[++i];
			if (strcmp(measure, "leven") == 0)
				ans->measure = LEVENSHTEIN;
			else if (strcmp(measure, "hamming") == 0)
				ans->measure = HAMMING;
			else
				return print_err("invalid measure option");
		}
		else
			return print_err("unknown option");
	}

	if (ans->seq1Path == NULL)
		return print_err("missing path for seq1");
	if (ans->seq1Len == 0)
		return print_err("missing length for seq1");
	if ((ans->infoPath == NULL) != (ans->infoLen == 0) )
		return print_err("repertiore path or repertoire count is missing in overlap mode");

	return SUCCESS;
}

/**
 * read and parse input csv file to Int3* and maybe int*
*/
int parse_input(char* path, Int3* seqOut, int* freqOut, int len, bool doubleCol) {
	FILE* file = fopen(path, "r");
	if (file == NULL)
		return print_err("file reading failed");

	const int BUFFER_SIZE = 50;
	char line[BUFFER_SIZE];
	int lineNumber = 0, inputCount = 0;

	// ignore header
	fgets(line, BUFFER_SIZE, file);

	while (fgets(line, BUFFER_SIZE, file)) {
		lineNumber++;
		if (strcmp(line, "\n") == 0 || strcmp(line, " \n") == 0)
			continue;

		Int3 newInt3 = str_encode(line);
		if (newInt3.entry[0] == 0) {
			fclose(file);
			return print_err_line("parsing error (only upper-cased amino acids with max length of 18 are allowed)", lineNumber);
		}
		seqOut[inputCount++] = newInt3;

		if (doubleCol) {
			char* line2 = strchr(line, ',');
			if (line2 == NULL) {
				fclose(file);
				return print_err_line("parsing error (comma expected)", lineNumber);
			}

			long int temp = strtol(line2);
			if (temp == 0 || temp > INT_MAX || temp < INT_MIN) {
				fclose(file);
				return print_err_line("parsing error (invalid number)", lineNumber);
			}
			freqOut[inputCount] = temp;
		}
	}

	fclose(file);
	if (inputCount != len)
		return print_err("input length doesn't match with the actual");

	return SUCCESS;
}

/**
 * read and parse info csv file to int*
*/
int parse_info(char* path, int* result, int len) {
	FILE* file = fopen(path, "r");
	if (file == NULL)
		return print_err("file reading failed");

	const int BUFFER_SIZE = 20;
	char line[BUFFER_SIZE];
	int lineNumber = 0, inputCount = 0;

	// ignore header
	fgets(line, BUFFER_SIZE, file);

	while (fgets(line, BUFFER_SIZE, file)) {
		lineNumber++;
		if (strcmp(line, "\n") == 0 || strcmp(line, " \n") == 0)
			continue;

		long int temp = strtol(line);
		if (temp == 0 || temp > INT_MAX || temp < INT_MIN) {
			fclose(file);
			return print_err_line("parsing error (invalid number)", lineNumber);
		}
		result[inputCount++] = temp;
	}

	fclose(file);
	if (inputCount != len)
		return print_err("info length doesn't match with the actual");

	return SUCCESS;
}

void null_handler(XTNOutput output) {
	totalOutputLen += output.len;
}

void file_handler_nn(XTNOutput output) {
	Int2 current;
	for (int i = 0; i < output.len; i++) {
		current = output.indexPairs[i];
		fprintf(outputFile, "%d %d %d\n", current.x, current.y , output.pairwiseDistances[i]);
	}
	totalOutputLen += output.len;
}

void file_handler_overlap(XTNOutput output) {
	Int2 current;
	for (int i = 0; i < output.len; i++) {
		current = output.indexPairs[i];
		fprintf(outputFile, "%d %d %'lu\n", current.x, current.y , output.pairwiseFrequencies[i]);
	}
	totalOutputLen += output.len;
}

int sum_check(int* repSizes, int seqLen, int n) {
	int sum = 0;
	for (int i = 0; i < n; i++)
		sum += repSizes[i];
	if (sum != seqLen)
		return print_err("total repertoires' size does not match sequence count");
	return SUCCESS;
}

int free_all(Int3* seq1, int* seqFreq, int* repSizes, int returnCode) {
	hipHostFree(seq1); gpuerr();
	if (seqFreq != NULL) {
		hipHostFree(seqFreq); gpuerr();
	}
	if (repSizes != NULL) {
		hipHostFree(repSizes); gpuerr();
	}
	return returnCode;
}

int main(int argc, char **argv) {
	XTNArgs args;
	int returnCode;
	Int3* seq1;
	int* seqFreq = NULL, *repSizes = NULL;

	// 1. parse command line arguments
	setlocale(LC_ALL, "");
	returnCode = parse_args(argc, argv, &args);
	verboseGlobal = args.verbose;
	if (returnCode != SUCCESS)
		return returnCode;

	// 2. read input
	bool overlapMode = args.infoPath != NULL;
	if (overlapMode) {
		hipHostMalloc(&repSizes, sizeof(int) * args.infoLen); gpuerr();
		returnCode = parse_info(args.infoPath, repSizes, args.infoLen);
		if (returnCode != SUCCESS)
			return free_all(seq1, seqFreq, repSizes, returnCode);
		hipHostMalloc(&seqFreq, sizeof(int) * args.seq1Len); gpuerr();
	}
	hipHostMalloc(&seq1, sizeof(Int3) * args.seq1Len); gpuerr();
	returnCode = parse_input(args.seq1Path, seq1, seqFreq, args.seq1Len, overlapMode);
	if (returnCode != SUCCESS)
		return free_all(seq1, seqFreq, repSizes, returnCode);
	returnCode = sum_check(repSizes, args.seq1Len, args.infoLen);
	if (returnCode != SUCCESS)
		return free_all(seq1, seqFreq, repSizes, returnCode);

	// 3. perform algorithm
	if (verboseGlobal)
		print_args(args);
	if (args.outputPath != NULL) {
		if (outputFile != NULL)
			return print_err("output file has already been allocated, possibly due to concurrency");
		outputFile = fopen(args.outputPath, "w");
		if (outputFile == NULL)
			return print_err("file reading failed");
		xtn_perform(args, seq1,
		            overlapMode ? file_handler_overlap : file_handler_nn);
		fclose(outputFile);
	} else {
		xtn_perform(args, seq1, null_handler);
	}

	printf("total output length: %'lu\n", totalOutputLen);
	return free_all(seq1, seqFreq, repSizes, SUCCESS);
}