#include "generate_combination.cu"
#include "cub.cu"
#include "kernel.cu"
#include "codec.cu"
#include "stream.cu"
#include <limits.h>

const int NUM_THREADS = 256;
const int HISTOGRAM_SIZE = 16; //TODO
const unsigned int UINT_MIN = 0;

//=====================================
// Private Functions
//=====================================

int cal_offsets(Int3* inputKeys, int* &inputOffsets, int* &outputLengths, int n, int* buffer) {
	// cal inputOffsets
	hipMalloc(&inputOffsets, sizeof(int)*n); gpuerr();
	unique_counts(inputKeys, inputOffsets, buffer, n); gpuerr();
	int nUnique = transfer_last_element(buffer, 1); gpuerr();

	// cal outputLengths
	int nUniqueBlock = divide_ceil(nUnique, NUM_THREADS);
	hipMalloc(&outputLengths, sizeof(int)*nUnique); gpuerr();
	cal_pair_len <<< nUniqueBlock, NUM_THREADS>>>(inputOffsets, outputLengths, nUnique); gpuerr();
	inclusive_sum(inputOffsets, nUnique); gpuerr();
	return nUnique;
}

int gen_pairs(int* input, int* inputOffsets, int* outputLengths, Int2* &output, int n) {
	int* outputOffsets;

	// cal outputOffsets
	hipMalloc(&outputOffsets, n * sizeof(int));
	inclusive_sum(outputLengths, outputOffsets, n); gpuerr();
	int outputLen = transfer_last_element(outputOffsets, n); gpuerr();

	//generate pairs
	int nBlock = divide_ceil(n, NUM_THREADS);
	hipMalloc(&output, sizeof(Int2)*outputLen); gpuerr();
	generate_pairs <<< nBlock, NUM_THREADS>>>(input, output,
	        inputOffsets, outputOffsets, n); gpuerr();

	hipFree(outputOffsets);
	return outputLen;
}

int gen_smaller_index(int* input, int* inputOffsets, int* outputLengths, int* &output, int n) {
	int* outputOffsets;

	// cal outputOffsets
	hipMalloc(&outputOffsets, n * sizeof(int));
	inclusive_sum(outputLengths, outputOffsets, n); gpuerr();
	int outputLen = transfer_last_element(outputOffsets, n); gpuerr();

	//generate pairs
	int nBlock = divide_ceil(n, NUM_THREADS);
	hipMalloc(&output, sizeof(Int2)*outputLen); gpuerr();
	generate_smaller_index <<< nBlock, NUM_THREADS>>>(input, output,
	        inputOffsets, outputOffsets, n); gpuerr();

	hipFree(outputOffsets);
	return outputLen;
}

int postprocessing(Int3* seq, Int2* input, int distance,
                   Int2* &pairOutput, char* &distanceOutput,
                   int n, int* buffer, int seqLen) {
	Int2* uniquePairs;
	char* uniqueDistances, *flags;

	// filter duplicate
	hipMalloc(&uniquePairs, sizeof(Int2)*n); gpuerr();
	sort_int2(input, n); gpuerr();
	unique(input, uniquePairs, buffer, n); gpuerr();

	// cal levenshtein
	int uniqueLen = transfer_last_element(buffer, 1); gpuerr();
	int byteRequirement = sizeof(char) * uniqueLen;
	int uniqueLenBlock = divide_ceil(uniqueLen, NUM_THREADS);
	hipMalloc(&flags, byteRequirement); gpuerr();
	hipMalloc(&uniqueDistances, byteRequirement); gpuerr();
	hipMalloc(&distanceOutput, byteRequirement); gpuerr();
	hipMalloc(&pairOutput, sizeof(Int2)*uniqueLen); gpuerr();
	cal_levenshtein <<< uniqueLenBlock, NUM_THREADS>>>(
	    seq, uniquePairs, distance, uniqueDistances, flags, uniqueLen, seqLen); gpuerr();

	// filter levenshtein
	double_flag(uniquePairs, uniqueDistances, flags, pairOutput,
	            distanceOutput, buffer, uniqueLen); gpuerr();
	_cudaFree(uniquePairs, uniqueDistances, flags); gpuerr();
	return transfer_last_element(buffer, 1);
}

void make_output(Int2* pairOut, char* distanceOut, size_t len, XTNOutput &output) {
	output.indexPairs = device_to_host(pairOut, len); gpuerr();
	output.pairwiseDistances = device_to_host(distanceOut, len); gpuerr();
	output.len = len;
}

void gen_next_chunk(Chunk<Int3> keyInput, Chunk<int> valueInput,
                    Chunk<Int3> &keyOutput, Chunk<int> &valueOutput,
                    int* valueOffsets, int offsetLen, int lowerbound, int* buffer) {
	char* flags;
	hipMalloc(&flags, sizeof(char)*valueInput.len); gpuerr();
	hipMemset(flags, 1, sizeof(char)*valueInput.len); gpuerr();
	int inputBlocks = divide_ceil(offsetLen, NUM_THREADS);

	flag_lowerbound <<< inputBlocks, NUM_THREADS>>>(
	    valueInput.ptr, valueOffsets, flags, lowerbound, offsetLen); gpuerr();
	double_flag(keyInput.ptr, valueInput.ptr, flags, keyOutput.ptr, valueOutput.ptr,
	            buffer, keyInput.len); gpuerr();

	int outputLen = transfer_last_element(buffer, 1); gpuerr();
	keyOutput.len = outputLen;
	valueOutput.len = outputLen;
	hipFree(flags); gpuerr();
}

int solve_next_bin(int* chunksizes, int start, int maxReadableSize, int n) {
	int ans = 0, len = 0;
	for (int i = start; i < n; i++) {
		int currentChunkSize = chunksizes[i];
		if (len + currentChunkSize > maxReadableSize)
			break;
		len += currentChunkSize;
		ans++;
	}
	return ans;
}

//=====================================
// Public Functions
//=====================================

int solve_bin_packing(int* histograms, int** &offsetOutput,
                      int maxProcessingExponent, int n, int nLevel, int* buffer) {
	int* rowIndex, *assignment, *output1d, *output1dPtr;

	int len2d = n * nLevel;
	int inputBlocks = divide_ceil(n, NUM_THREADS);
	int inputBlocks2 = divide_ceil(nLevel, NUM_THREADS);
	hipMalloc(&rowIndex, sizeof(int) * len2d); gpuerr();
	hipMalloc(&assignment, sizeof(int) * len2d); gpuerr();
	hipMalloc(&output1d, sizeof(int) * len2d); gpuerr();
	hipHostMalloc(&offsetOutput, sizeof(int*) * n); gpuerr();

	//solve bin packing
	make_row_index <<< inputBlocks, NUM_THREADS>>>(rowIndex, n, nLevel);
	inclusive_sum_by_key(rowIndex, histograms, len2d); gpuerr();
	gen_assignment <<< inputBlocks2, NUM_THREADS >>>(
	    histograms, assignment, maxProcessingExponent, n, nLevel); gpuerr();
	max_by_key(assignment, histograms, output1d, buffer, len2d); gpuerr();

	//make output
	int outputLen = transfer_last_element(buffer, 1); gpuerr();
	if (outputLen % n != 0)
		print_err("bin_packing outputLen is not divisible by inputLen");
	int offsetLen = outputLen / n;
	output1dPtr = output1d;
	for (int i = 0; i < n; i++) {
		offsetOutput[i] = device_to_host(output1dPtr, offsetLen); gpuerr();
		output1dPtr += offsetLen;
	}

	_cudaFree(rowIndex, assignment, output1d); gpuerr();
	return offsetLen;
}

void stream_handler1(Chunk<Int3> input, Int3* &deletionsOutput, int* &indexOutput,
                     int* &histogramOutput, int &outputLen, int distance) {
	// boilerplate
	int *combinationOffsets;
	unsigned int *histogramValue;
	int inputBlocks = divide_ceil(input.len, NUM_THREADS);

	// cal combinationOffsets
	hipMalloc(&combinationOffsets, sizeof(int)*input.len);	gpuerr();
	cal_combination_len <<< inputBlocks, NUM_THREADS >>>(
	    input.ptr, distance, combinationOffsets, input.len); gpuerr();
	inclusive_sum(combinationOffsets, input.len); gpuerr();
	outputLen = transfer_last_element(combinationOffsets, input.len); gpuerr();

	// generate combinations
	hipMalloc(&deletionsOutput, sizeof(Int3)*outputLen); gpuerr();
	hipMalloc(&indexOutput, sizeof(int)*outputLen); gpuerr();
	gen_combination <<< inputBlocks, NUM_THREADS >>> (
	    input.ptr, combinationOffsets, distance,
	    deletionsOutput, indexOutput, input.len); gpuerr();

	// generate histogram
	int outputBlocks = divide_ceil(outputLen , NUM_THREADS);
	hipMalloc(&histogramValue, sizeof(unsigned int)*outputLen);
	hipMalloc(&histogramOutput, sizeof(int)*HISTOGRAM_SIZE);
	select_int3 <<< outputBlocks, NUM_THREADS>>>(
	    deletionsOutput, histogramValue, outputLen);
	cal_histogram(histogramValue, histogramOutput, HISTOGRAM_SIZE, UINT_MIN, UINT_MAX, outputLen);
	sort_key_values(deletionsOutput, indexOutput, outputLen);

	// boilerplate
	_cudaFree(combinationOffsets, histogramValue); gpuerr();
}

void stream_handler2(Chunk<Int3> &keyInOut, Chunk<int> &valueInOut, int* &histogramOutput,
                     int distance, int seqLen, int memoryConstraint, int* buffer) {
	int* inputOffsets, *valueLengths, *histogram, *indexes, *valueLengthsHost;

	printf("1\n");
	sort_key_values(keyInOut.ptr, valueInOut.ptr, keyInOut.len); gpuerr();
	printf("2\n");
	int offsetLen =
	    cal_offsets(keyInOut.ptr, inputOffsets, valueLengths, keyInOut.len, buffer); gpuerr();
	printf("3\n");

	int start = 0, nChunk;
	int* inputOffsetsPtr = inputOffsets, *valueLengthsPtr = valueLengths;
	int nBlock = divide_ceil(HISTOGRAM_SIZE , NUM_THREADS);
	valueLengthsHost = device_to_host(valueLengths, offsetLen);
	printf("4\n");

	//histogram loop
	while ((nChunk = solve_next_bin(valueLengthsHost, start, memoryConstraint, offsetLen)) > 0) {
		printf("5\n");
		int chunkLen = gen_smaller_index(valueInOut.ptr, inputOffsetsPtr, valueLengthsPtr, indexes, nChunk);
		printf("6\n");
		cal_histogram(indexes, histogram, HISTOGRAM_SIZE , 0, seqLen, chunkLen);
		printf("7\n");
		vector_add <<< nBlock, NUM_THREADS>>>(histogramOutput, histogram, HISTOGRAM_SIZE);
		printf("8\n");

		start += nChunk;
		inputOffsetsPtr += nChunk;
		valueLengthsPtr += nChunk;
		_cudaFree(indexes, histogram); gpuerr();
		printf("9\n");
	}
	_cudaFree(inputOffsets, valueLengths); gpuerr();
	hipHostFree(valueLengthsHost); gpuerr();
}

// void stream_handler3(Chunk<Int3> keyInput, Chunk<int> valueInput,
//                      Chunk<Int3> &keyOutput, Chunk<int> &valueOutput, Int2* &pairOutput,
//                      int* &histogramOutput, int lowerbound, int* buffer) {
// 	int* combinationValueOffsets, *pairOffsets;
// 	int offsetLen =
// 	    cal_offsets(keyInput.ptr, valueInput.ptr, combinationValueOffsets,
// 	                pairOffsets, keyInput.len, buffer);
// 	int pairLen =
// 	    gen_pairs(valueInput.ptr, combinationValueOffsets,
// 	              pairOffsets, pairOutput, offsetLen, buffer);

// 	// generate histogram
// 	// take lower bound into account

// 	gen_next_chunk(keyInput, valueInput, keyOutput, valueOutput,
// 	               combinationValueOffsets, offsetLen, lowerbound, buffer);
// 	_cudaFree(combinationValueOffsets, pairOffsets);
// }

void stream_handler4(Chunk<Int2> pairInput, XTNOutput &output, Int3* seq1,
                     int seq1Len, int distance, int* buffer) {
	Int2* pairOut;
	char* distanceOut;
	int outputLen =
	    postprocessing(seq1, pairInput.ptr, distance, pairOut, distanceOut,
	                   pairInput.len, buffer, seq1Len);

	make_output(pairOut, distanceOut, outputLen, output);
	_cudaFree(pairOut, distanceOut);
}