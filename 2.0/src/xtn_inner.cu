#include "generate_combination.cu"
#include "cub.cu"
#include "kernel.cu"
#include "codec.cu"
#include "stream.cu"
#include <limits.h>

const int NUM_THREADS = 256;
const int HISTOGRAM_SIZE = 16; //TODO
const unsigned int UINT_MIN = 0;

//=====================================
// Private Functions
//=====================================

int NUM_BLOCK(int len) {
	return divide_ceil(len, NUM_THREADS);
}

int cal_offsets(Int3* inputKeys, int* &inputOffsets, int* &outputLengths, int n, int* buffer) {
	// cal inputOffsets
	hipMalloc(&inputOffsets, sizeof(int)*n); gpuerr();
	unique_counts(inputKeys, inputOffsets, buffer, n); gpuerr();
	int nUnique = transfer_last_element(buffer, 1); gpuerr();

	// cal outputLengths
	hipMalloc(&outputLengths, sizeof(int)*nUnique); gpuerr();
	cal_pair_len <<< NUM_BLOCK(nUnique), NUM_THREADS>>>(inputOffsets, outputLengths, nUnique); gpuerr();
	inclusive_sum(inputOffsets, nUnique); gpuerr();
	return nUnique;
}

int cal_offsets_lowerbound(Int3* inputKeys, int* inputValues, int* &inputOffsets,
                           int* &outputLengths, int lowerbound, int n, int* buffer) {
	// cal inputOffsets
	hipMalloc(&inputOffsets, sizeof(int)*n); gpuerr();
	unique_counts(inputKeys, inputOffsets, buffer, n); gpuerr();
	int nUnique = transfer_last_element(buffer, 1); gpuerr();
	inclusive_sum(inputOffsets, nUnique); gpuerr();

	// cal outputLengths
	hipMalloc(&outputLengths, sizeof(int)*nUnique); gpuerr();
	cal_pair_len_lowerbound <<< NUM_BLOCK(nUnique), NUM_THREADS>>>(
	    inputValues, inputOffsets, outputLengths, lowerbound, nUnique); gpuerr();
	return nUnique;
}

int gen_pairs(int* input, int* inputOffsets, int* outputLengths, Int2* &output,
              int* &lesserIndex, int lowerbound, int n) {
	int* outputOffsets;

	// cal outputOffsets
	hipMalloc(&outputOffsets, n * sizeof(int)); gpuerr();
	inclusive_sum(outputLengths, outputOffsets, n); gpuerr();
	int outputLen = transfer_last_element(outputOffsets, n); gpuerr();

	//generate pairs
	hipMalloc(&output, sizeof(Int2)*outputLen); gpuerr();
	hipMalloc(&lesserIndex, sizeof(int)*outputLen); gpuerr();
	generate_pairs <<< NUM_BLOCK(n), NUM_THREADS>>>(input, output,
	        inputOffsets, outputOffsets, lesserIndex, lowerbound, n); gpuerr();

	hipFree(outputOffsets); gpuerr();
	return outputLen;
}

int gen_smaller_index(int* input, int* inputOffsets, int* outputLengths, int* &output, int n) {
	int* outputOffsets;

	// cal outputOffsets
	hipMalloc(&outputOffsets, n * sizeof(int)); gpuerr();
	inclusive_sum(outputLengths, outputOffsets, n); gpuerr();
	int outputLen = transfer_last_element(outputOffsets, n); gpuerr();

	//generate pairs
	hipMalloc(&output, sizeof(int)*outputLen); gpuerr();
	generate_smaller_index <<< NUM_BLOCK(n), NUM_THREADS>>>(input, output,
	        inputOffsets, outputOffsets, n); gpuerr();

	hipFree(outputOffsets); gpuerr();
	return outputLen;
}

int postprocessing(Int3* seq, Int2* input, int distance,
                   Int2* &pairOutput, char* &distanceOutput,
                   int n, int* buffer, int seqLen) {
	Int2* uniquePairs;
	char* uniqueDistances, *flags;

	// filter duplicate
	hipMalloc(&uniquePairs, sizeof(Int2)*n); gpuerr();
	sort_int2(input, n); gpuerr();
	unique(input, uniquePairs, buffer, n); gpuerr();

	// cal levenshtein
	int uniqueLen = transfer_last_element(buffer, 1); gpuerr();
	int byteRequirement = sizeof(char) * uniqueLen;
	hipMalloc(&flags, byteRequirement); gpuerr();
	hipMalloc(&uniqueDistances, byteRequirement); gpuerr();
	hipMalloc(&distanceOutput, byteRequirement); gpuerr();
	hipMalloc(&pairOutput, sizeof(Int2)*uniqueLen); gpuerr();
	cal_levenshtein <<< NUM_BLOCK(uniqueLen), NUM_THREADS>>>(
	    seq, uniquePairs, distance, uniqueDistances, flags, uniqueLen, seqLen); gpuerr();

	// filter levenshtein
	double_flag(uniquePairs, uniqueDistances, flags, pairOutput,
	            distanceOutput, buffer, uniqueLen); gpuerr();
	_cudaFree(uniquePairs, uniqueDistances, flags); gpuerr();
	int outputLen = transfer_last_element(buffer, 1); gpuerr();
	return outputLen;
}

void make_output(Int2* pairOut, char* distanceOut, size_t len, XTNOutput &output) {
	output.indexPairs = device_to_host(pairOut, len); gpuerr();
	output.pairwiseDistances = device_to_host(distanceOut, len); gpuerr();
	output.len = len;
}

void gen_next_chunk(Chunk<Int3> &keyInOut, Chunk<int> &valueInOut,
                    int* valueOffsets, int offsetLen, int lowerbound, int* buffer) {
	char* flags;
	Int3* keyOut;
	int* valueOut;

	hipMalloc(&flags, sizeof(char)*valueInOut.len); gpuerr();
	hipMemset(flags, 1, sizeof(char)*valueInOut.len); gpuerr();
	hipMalloc(&keyOut, sizeof(Int3)*valueInOut.len); gpuerr();
	hipMalloc(&valueOut, sizeof(int)*valueInOut.len); gpuerr();

	flag_lowerbound <<< NUM_BLOCK(offsetLen), NUM_THREADS>>>(
	    valueInOut.ptr, valueOffsets, flags, lowerbound, offsetLen); gpuerr();
	double_flag(keyInOut.ptr, valueInOut.ptr, flags, keyOut, valueOut,
	            buffer, valueInOut.len); gpuerr();

	int outputLen = transfer_last_element(buffer, 1); gpuerr();
	_cudaFree(flags, keyInOut.ptr, valueInOut.ptr); gpuerr();
	keyInOut.ptr = keyOut;
	keyInOut.len = outputLen;
	valueInOut.ptr = valueOut;
	valueInOut.len = outputLen;
}

int solve_next_bin(int* chunksizes, int start, int maxReadableSize, int n) {
	int ans = 0, len = 0;
	for (int i = start; i < n; i++) {
		int currentChunkSize = chunksizes[i];
		if (len + currentChunkSize > maxReadableSize)
			break;
		len += currentChunkSize;
		ans++;
	}
	return ans;
}

//=====================================
// Public Functions
//=====================================

int solve_bin_packing(int* histograms, int** &offsetOutput,
                      int n, int nLevel, int* buffer, MemoryContext ctx) {
	int* rowIndex, *assignment, *output1d, *output1dPtr;

	int len2d = n * nLevel;
	hipMalloc(&rowIndex, sizeof(int) * len2d); gpuerr();
	hipMalloc(&assignment, sizeof(int) * len2d); gpuerr();
	hipMalloc(&output1d, sizeof(int) * len2d); gpuerr();
	hipHostMalloc(&offsetOutput, sizeof(int*) * n); gpuerr();

	//solve bin packing
	make_row_index <<< NUM_BLOCK(n), NUM_THREADS>>>(rowIndex, n, nLevel);
	inclusive_sum_by_key(rowIndex, histograms, len2d); gpuerr();
	gen_assignment <<< NUM_BLOCK(nLevel), NUM_THREADS >>>(
	    histograms, assignment, ctx.maxThroughputExponent, n, nLevel); gpuerr();
	max_by_key(assignment, histograms, output1d, buffer, len2d); gpuerr();

	//make output
	int outputLen = transfer_last_element(buffer, 1); gpuerr();
	if (outputLen % n != 0)
		print_err("bin_packing outputLen is not divisible by inputLen");
	int offsetLen = outputLen / n;
	output1dPtr = output1d;
	for (int i = 0; i < n; i++) {
		offsetOutput[i] = device_to_host(output1dPtr, offsetLen); gpuerr();
		output1dPtr += offsetLen;
	}

	_cudaFree(rowIndex, assignment, output1d); gpuerr();
	return offsetLen;
}

void stream_handler1(Chunk<Int3> input, Int3* &deletionsOutput, int* &indexOutput,
                     int* &histogramOutput, int &outputLen, int distance, MemoryContext ctx) {
	int *combinationOffsets;
	unsigned int *histogramValue;

	// cal combinationOffsets
	hipMalloc(&combinationOffsets, sizeof(int)*input.len);	gpuerr();
	cal_combination_len <<< NUM_BLOCK(input.len), NUM_THREADS >>>(
	    input.ptr, distance, combinationOffsets, input.len); gpuerr();
	inclusive_sum(combinationOffsets, input.len); gpuerr();
	outputLen = transfer_last_element(combinationOffsets, input.len); gpuerr();

	// generate combinations
	hipMalloc(&deletionsOutput, sizeof(Int3)*outputLen); gpuerr();
	hipMalloc(&indexOutput, sizeof(int)*outputLen); gpuerr();
	hipMalloc(&histogramValue, sizeof(unsigned int)*outputLen); gpuerr();
	gen_combination <<< NUM_BLOCK(input.len), NUM_THREADS >>> (
	    input.ptr, combinationOffsets, distance,
	    deletionsOutput, indexOutput, histogramValue, input.len); gpuerr();

	// generate histogram
	hipMalloc(&histogramOutput, sizeof(int)*ctx.histogramSize); gpuerr();
	cal_histogram(histogramValue, histogramOutput, ctx.histogramSize, UINT_MIN, UINT_MAX, outputLen); gpuerr();
	sort_key_values(deletionsOutput, indexOutput, outputLen); gpuerr();

	_cudaFree(combinationOffsets, histogramValue); gpuerr();
}

void stream_handler2(Chunk<Int3> &keyInOut, Chunk<int> &valueInOut, int* &histogramOutput,
                     int distance, int seqLen, int* buffer, MemoryContext ctx) {
	int* inputOffsets, *valueLengths, *histogram, *indexes, *valueLengthsHost;

	sort_key_values(keyInOut.ptr, valueInOut.ptr, keyInOut.len); gpuerr();
	int offsetLen =
	    cal_offsets(keyInOut.ptr, inputOffsets, valueLengths, keyInOut.len, buffer);

	int start = 0, nChunk;
	int* inputOffsetsPtr = inputOffsets, *valueLengthsPtr = valueLengths;
	valueLengthsHost = device_to_host(valueLengths, offsetLen); gpuerr();
	int nBlock = divide_ceil(ctx.histogramSize, NUM_THREADS);
	hipMalloc(&histogram, sizeof(int)*ctx.histogramSize); gpuerr();

	//histogram loop
	while ((nChunk = solve_next_bin(valueLengthsHost, start, ctx.maxThroughput, offsetLen)) > 0) {
		int chunkLen = gen_smaller_index(valueInOut.ptr, inputOffsetsPtr, valueLengthsPtr, indexes, nChunk);
		cal_histogram(indexes, histogram, ctx.histogramSize , 0, seqLen, chunkLen); gpuerr();
		vector_add <<< nBlock, NUM_THREADS>>>(histogramOutput, histogram, ctx.histogramSize); gpuerr();

		start += nChunk;
		inputOffsetsPtr += nChunk;
		valueLengthsPtr += nChunk;
		hipFree(indexes); gpuerr();
	}

	_cudaFree(inputOffsets, valueLengths, histogram); gpuerr();
	hipHostFree(valueLengthsHost); gpuerr();
}

void stream_handler3(Chunk<Int3> &keyInOut, Chunk<int> &valueInOut, D2Stream<Int2> &pairOut,
                     int* &histogramOutput, int lowerbound, int seqLen, int* buffer, MemoryContext ctx) {
	int* inputOffsets, *valueLengths, *valueLengthsHost, *histogram, *lesserIndex;
	Int2* pairOutput;

	int offsetLen = cal_offsets_lowerbound(
	                    keyInOut.ptr, valueInOut.ptr, inputOffsets,
	                    valueLengths, lowerbound, keyInOut.len, buffer);

	int start = 0, nChunk;
	int* inputOffsetsPtr = inputOffsets, *valueLengthsPtr = valueLengths;
	valueLengthsHost = device_to_host(valueLengths, offsetLen); gpuerr();
	int nBlock = divide_ceil(ctx.histogramSize, NUM_THREADS);
	hipMalloc(&histogram, sizeof(int)*ctx.histogramSize); gpuerr();

	// generate pairs
	while ((nChunk = solve_next_bin(valueLengthsHost, start, ctx.maxThroughput, offsetLen)) > 0) {
		int chunkLen = gen_pairs(valueInOut.ptr, inputOffsetsPtr, valueLengthsPtr,
		                         pairOutput, lesserIndex, lowerbound, nChunk);
		pairOut.write(pairOutput, chunkLen);
		cal_histogram(lesserIndex, histogram, ctx.histogramSize , 0, seqLen, chunkLen); gpuerr();
		vector_add <<< nBlock, NUM_THREADS>>>(histogramOutput, histogram, ctx.histogramSize); gpuerr();

		start += nChunk;
		inputOffsetsPtr += nChunk;
		valueLengthsPtr += nChunk;
		_cudaFree(pairOutput, lesserIndex); gpuerr();
	}

	gen_next_chunk(keyInOut, valueInOut, inputOffsets, offsetLen, lowerbound, buffer);

	_cudaFree(inputOffsets, valueLengths, histogram); gpuerr();
	hipHostFree(valueLengthsHost); gpuerr();
}

void stream_handler4(Chunk<Int2> pairInput, XTNOutput & output, Int3 * seq1,
                     int seq1Len, int distance, int* buffer) {
	Int2* pairOut;
	char* distanceOut;
	int outputLen =
	    postprocessing(seq1, pairInput.ptr, distance, pairOut, distanceOut,
	                   pairInput.len, buffer, seq1Len);

	make_output(pairOut, distanceOut, outputLen, output);
	_cudaFree(pairOut, distanceOut); gpuerr();
}