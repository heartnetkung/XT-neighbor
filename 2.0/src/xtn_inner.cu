#include "generate_combination.cu"
#include "cub.cu"
#include "kernel.cu"
#include "codec.cu"
#include "stream.cu"

const int NUM_THREADS = 256;

int cal_offsets(Int3* inputKeys, int* inputValues, int* &inputOffsets, int* &outputOffsets, int n, int* buffer) {
	// cal valueOffsets
	hipMalloc(&inputOffsets, sizeof(int)*n); gpuerr();
	// assume sorted
	// sort_key_values(inputKeys, inputValues, n); gpuerr();
	unique_counts(inputKeys, inputOffsets, buffer, n); gpuerr();

	// cal pairOffsets
	int nUnique = transfer_last_element(buffer, 1); gpuerr();
	int nUniqueBlock = divideCeil(nUnique, NUM_THREADS);
	hipMalloc(&outputOffsets, sizeof(int)*nUnique); gpuerr();
	cal_pair_len <<< nUniqueBlock, NUM_THREADS>>>(inputOffsets, outputOffsets, nUnique); gpuerr();
	inclusive_sum(inputOffsets, nUnique); gpuerr();
	inclusive_sum(outputOffsets, nUnique); gpuerr();
	return nUnique;
}

int gen_pairs(int* input, int* inputOffsets, int* outputOffsets, Int2* &output, int n, int* buffer) {
	// generate pairs
	int outputLen = transfer_last_element(outputOffsets, n); gpuerr();
	int nBlock = divideCeil(n, NUM_THREADS);
	hipMalloc(&output, sizeof(Int2)*outputLen); gpuerr();
	generate_pairs <<< nBlock, NUM_THREADS>>>(input, output,
	        inputOffsets, outputOffsets, n); gpuerr();
	return outputLen;
}

int postprocessing(Int3* seq, Int2* input, int distance,
                   Int2* &pairOutput, char* &distanceOutput,
                   int n, int* buffer, int seqLen) {
	Int2* uniquePairs;
	char* uniqueDistances, *flags;

	// filter duplicate
	hipMalloc(&uniquePairs, sizeof(Int2)*n); gpuerr();
	sort_int2(input, n); gpuerr();
	unique(input, uniquePairs, buffer, n); gpuerr();

	// cal levenshtein
	int uniqueLen = transfer_last_element(buffer, 1); gpuerr();
	int byteRequirement = sizeof(char) * uniqueLen;
	int uniqueLenBlock = divideCeil(uniqueLen, NUM_THREADS);
	hipMalloc(&flags, byteRequirement); gpuerr();
	hipMalloc(&uniqueDistances, byteRequirement); gpuerr();
	hipMalloc(&distanceOutput, byteRequirement); gpuerr();
	hipMalloc(&pairOutput, sizeof(Int2)*uniqueLen); gpuerr();
	cal_levenshtein <<< uniqueLenBlock, NUM_THREADS>>>(
	    seq, uniquePairs, distance, uniqueDistances, flags, uniqueLen, seqLen); gpuerr();

	// filter levenshtein
	double_flag(uniquePairs, uniqueDistances, flags, pairOutput,
	            distanceOutput, buffer, uniqueLen); gpuerr();
	_cudaFree(uniquePairs, uniqueDistances, flags); gpuerr();
	return transfer_last_element(buffer, 1);
}

void make_output(Int2* pairOut, char* distanceOut, size_t len, XTNOutput &output) {
	output.indexPairs = device_to_host(pairOut, len); gpuerr();
	output.pairwiseDistances = device_to_host(distanceOut, len); gpuerr();
	output.len = len;
}

void gen_next_chunk(Chunk<Int3> keyInput, Chunk<int> valueInput,
                    Chunk<Int3> &keyOutput, Chunk<int> &valueOutput,
                    int* valueOffsets, int offsetLen, int lowerbound, int* buffer) {
	char* flags;
	hipMalloc(&flags, sizeof(char)*valueInput.len); gpuerr();
	hipMemset(flags, 1, valueInput.len); gpuerr();
	int inputBlocks = divideCeil(offsetLen, NUM_THREADS);

	flag_lowerbound <<< inputBlocks, NUM_THREADS>>>(
	    valueInput.ptr, valueOffsets, flags, lowerbound, offsetLen); gpuerr();
	double_flag(keyInput.ptr, valueInput.ptr, flags, keyOutput.ptr, valueOutput.ptr,
	            buffer, keyInput.len); gpuerr();

	int outputLen = transfer_last_element(buffer, 1); gpuerr();
	keyOutput.len = outputLen;
	valueOutput.len = outputLen;
	hipFree(flags);
}


void stream_handler1(Chunk<Int3> input, Chunk<Int3> &output1,
                     Chunk<int> &output2, int distance) {
	int *combinationOffsets;
	int n = input.len;
	int inputBlocks = divideCeil(n, NUM_THREADS);

	// cal combinationOffsets
	hipMalloc((void**)&combinationOffsets, sizeof(int)*n);	gpuerr();
	cal_combination_len <<< inputBlocks, NUM_THREADS >>>(
	    input.ptr, distance, combinationOffsets, n); gpuerr();
	inclusive_sum(combinationOffsets, n); gpuerr();
	int outputLen = transfer_last_element(combinationOffsets, n); gpuerr();

	// generate combinations
	hipMalloc(&output1.ptr, sizeof(Int3)*outputLen); gpuerr();
	hipMalloc(&output2.ptr, sizeof(int)*outputLen); gpuerr();
	gen_combination <<< inputBlocks, NUM_THREADS >>> (
	    input.ptr, combinationOffsets, distance, output1.ptr, output2.ptr, n); gpuerr();

	hipFree(combinationOffsets); gpuerr();
	output1.len = outputLen;
	output2.len = outputLen;
}

void stream_handler2() {

}

void stream_handler3(Chunk<Int3> keyInput, Chunk<int> valueInput,
                     Chunk<Int3> &keyOutput, Chunk<int> &valueOutput,
                     XTNOutput &output, Int3* seq1, int seq1Len,
                     int distance, int lowerbound, int* buffer) {

	int* combinationValueOffsets, *pairOffsets;
	int offsetLen =
	    cal_offsets(keyInput.ptr, valueInput.ptr, combinationValueOffsets,
	                pairOffsets, keyInput.len, buffer);

	Int2* pairs;
	int pairLen = gen_pairs(valueInput.ptr, combinationValueOffsets,
	                        pairOffsets, pairs, offsetLen, buffer);

	Int2* pairOut;
	char* distanceOut;
	int outputLen = postprocessing(seq1, pairs, distance,
	                               pairOut, distanceOut,
	                               pairLen, buffer, seq1Len);

	make_output(pairOut, distanceOut, outputLen, output);
	gen_next_chunk(keyInput, valueInput, keyOutput, valueOutput,
	               combinationValueOffsets, offsetLen, lowerbound, buffer);
	_cudaFree(combinationValueOffsets, pairOffsets, pairs, pairOut, distanceOut);
}