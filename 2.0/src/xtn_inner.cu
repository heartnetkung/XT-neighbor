#include "generate_combination.cu"
#include "cub.cu"
#include "kernel.cu"
#include "codec.cu"
#include "stream.cu"

const int NUM_THREADS = 256;

int cal_offsets(Int3* inputKeys, int* inputValues, int* &inputOffsets, int* &outputOffsets, int n, int* buffer) {
	// cal valueOffsets
	hipMalloc(&inputOffsets, sizeof(int)*n); gpuerr();
	sort_key_values(inputKeys, inputValues, n); gpuerr();
	unique_counts(inputKeys, inputOffsets, buffer, n); gpuerr();

	// cal pairOffsets
	int nUnique = transfer_last_element(buffer, 1); gpuerr();
	int nUniqueBlock = divideCeil(nUnique, NUM_THREADS);
	hipMalloc(&outputOffsets, sizeof(int)*nUnique); gpuerr();
	cal_pair_len <<< nUniqueBlock, NUM_THREADS>>>(inputOffsets, outputOffsets, nUnique); gpuerr();
	inclusive_sum(inputOffsets, nUnique); gpuerr();
	inclusive_sum(outputOffsets, nUnique); gpuerr();
	return nUnique;
}

int gen_pairs(int* input, int* inputOffsets, int* outputOffsets, Int2* &output, int n, int* buffer) {
	// generate pairs
	int outputLen = transfer_last_element(outputOffsets, n); gpuerr();
	int nBlock = divideCeil(n, NUM_THREADS);
	hipMalloc(&output, sizeof(Int2)*outputLen); gpuerr();
	generate_pairs <<< nBlock, NUM_THREADS>>>(input, output,
	        inputOffsets, outputOffsets, n); gpuerr();
	return outputLen;
}

int postprocessing(Int3* seq, Int2* input, int distance,
                   Int2* &pairOutput, char* &distanceOutput,
                   int n, int* buffer, int seqLen) {
	Int2* uniquePairs;
	char* uniqueDistances, *flags;

	// filter duplicate
	hipMalloc(&uniquePairs, sizeof(Int2)*n); gpuerr();
	sort_int2(input, n); gpuerr();
	unique(input, uniquePairs, buffer, n); gpuerr();

	// cal levenshtein
	int uniqueLen = transfer_last_element(buffer, 1);
	int byteRequirement = sizeof(char) * uniqueLen;
	int uniqueLenBlock = divideCeil(uniqueLen, NUM_THREADS);
	hipMalloc(&flags, byteRequirement); gpuerr();
	hipMalloc(&uniqueDistances, byteRequirement); gpuerr();
	hipMalloc(&distanceOutput, byteRequirement); gpuerr();
	hipMalloc(&pairOutput, sizeof(Int2)*uniqueLen); gpuerr();
	cal_levenshtein <<< uniqueLenBlock, NUM_THREADS>>>(
	    seq, uniquePairs, distance, uniqueDistances, flags, uniqueLen, seqLen); gpuerr();

	// filter levenshtein
	double_flag(uniquePairs, uniqueDistances, flags, pairOutput,
	            distanceOutput, buffer, uniqueLen); gpuerr();
	_cudaFree(uniquePairs, uniqueDistances, flags); gpuerr();
	return transfer_last_element(buffer, 1);
}


void stream_handler1(Chunk<Int3> input, Chunk<Int3> &output1,
                     Chunk<int> &output2, int distance) {
	int *combinationOffsets;
	int n = input.len;
	int inputBlocks = divideCeil(n, NUM_THREADS);

	// cal combinationOffsets
	hipMalloc((void**)&combinationOffsets, sizeof(int)*n);	gpuerr();
	cal_combination_len <<< inputBlocks, NUM_THREADS >>>(
	    input.ptr, distance, combinationOffsets, n); gpuerr();
	inclusive_sum(combinationOffsets, n); gpuerr();
	int outputLen = transfer_last_element(combinationOffsets, n); gpuerr();

	// generate combinations
	hipMalloc(&output1.ptr, sizeof(Int3)*outputLen); gpuerr();
	hipMalloc(&output2.ptr, sizeof(int)*outputLen); gpuerr();
	gen_combination <<< inputBlocks, NUM_THREADS >>> (
	    input.ptr, combinationOffsets, distance, output1.ptr, output2.ptr, n); gpuerr();

	hipFree(combinationOffsets); gpuerr();
	output1.len = outputLen;
	output2.len = outputLen;
}

void stream_handler2() {

}

void stream_handler3() {

}