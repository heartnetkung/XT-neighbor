#include "generate_combination.cu"
#include "cub.cu"
#include "kernel.cu"
#include "codec.cu"
#include "stream.cu"
#include <limits.h>

/**
 * @file
 * @brief The core algorithm on the high-level abstraction
 * concerning data manupulation operations in all 4 stream.
 */

const int NUM_THREADS = 256;
const unsigned int UINT_MIN = 0;

//=====================================
// Private Functions
//=====================================

int NUM_BLOCK(int len) {
	return divide_ceil(len, NUM_THREADS);
}

int cal_offsets(Int3* inputKeys, int* &inputOffsets, int* &outputLengths, int n, int* buffer) {
	// cal inputOffsets
	hipMalloc(&inputOffsets, sizeof(int)*n); gpuerr();
	unique_counts(inputKeys, inputOffsets, buffer, n); gpuerr();
	int nUnique = transfer_last_element(buffer, 1); gpuerr();

	// cal outputLengths
	hipMalloc(&outputLengths, sizeof(int)*nUnique); gpuerr();
	cal_pair_len <<< NUM_BLOCK(nUnique), NUM_THREADS>>>(inputOffsets, outputLengths, nUnique); gpuerr();
	inclusive_sum(inputOffsets, nUnique); gpuerr();
	return nUnique;
}

int cal_offsets_lowerbound(Int3* inputKeys, int* inputValues, int* &inputOffsets,
                           int* &outputLengths, int lowerbound, int n, int* buffer) {
	// cal inputOffsets
	hipMalloc(&inputOffsets, sizeof(int)*n); gpuerr();
	unique_counts(inputKeys, inputOffsets, buffer, n); gpuerr();
	int nUnique = transfer_last_element(buffer, 1); gpuerr();
	inclusive_sum(inputOffsets, nUnique); gpuerr();

	// cal outputLengths
	hipMalloc(&outputLengths, sizeof(int)*nUnique); gpuerr();
	cal_pair_len_lowerbound <<< NUM_BLOCK(nUnique), NUM_THREADS>>>(
	    inputValues, inputOffsets, outputLengths, lowerbound, nUnique); gpuerr();
	return nUnique;
}

int gen_pairs(int* input, int* inputOffsets, int* outputLengths, Int2* &output,
              int* &lesserIndex, int lowerbound, int carry, int n) {
	int* outputOffsets;

	// cal outputOffsets
	hipMalloc(&outputOffsets, n * sizeof(int)); gpuerr();
	inclusive_sum(outputLengths, outputOffsets, n); gpuerr();
	int outputLen = transfer_last_element(outputOffsets, n); gpuerr();

	//generate pairs
	hipMalloc(&output, sizeof(Int2)*outputLen); gpuerr();
	hipMalloc(&lesserIndex, sizeof(int)*outputLen); gpuerr();
	generate_pairs <<< NUM_BLOCK(n), NUM_THREADS>>>(input, output,
	        inputOffsets, outputOffsets, lesserIndex, lowerbound, carry, n); gpuerr();
	sort_int2(output, outputLen);

	print_gpu_memory();
	hipFree(outputOffsets); gpuerr();
	return outputLen;
}

int gen_smaller_index(int* input, int* inputOffsets, int* outputLengths,
                      int* &output, int carry, int n) {
	int* outputOffsets;

	// cal outputOffsets
	hipMalloc(&outputOffsets, n * sizeof(int)); gpuerr();
	inclusive_sum(outputLengths, outputOffsets, n); gpuerr();
	int outputLen = transfer_last_element(outputOffsets, n); gpuerr();

	//generate pairs
	hipMalloc(&output, sizeof(int)*outputLen); gpuerr();
	generate_smaller_index <<< NUM_BLOCK(n), NUM_THREADS>>>(input, output,
	        inputOffsets, outputOffsets, carry, n); gpuerr();

	print_gpu_memory();
	hipFree(outputOffsets); gpuerr();
	return outputLen;
}

int postprocessing(Int3* seq, Int2* input, int distance,
                   Int2* &pairOutput, char* &distanceOutput,
                   int n, int* buffer, int seqLen) {
	Int2* uniquePairs;
	char* uniqueDistances, *flags;

	// filter duplicate
	hipMalloc(&uniquePairs, sizeof(Int2)*n); gpuerr();
	sort_int2(input, n); gpuerr();
	unique(input, uniquePairs, buffer, n); gpuerr();


	// cal levenshtein
	int uniqueLen = transfer_last_element(buffer, 1); gpuerr();
	print_int2_arr(uniquePairs, 20);
	print_int2_arr(uniquePairs + uniqueLen - 20, 20);
	printf("uniqueLen %'d \n", uniqueLen);
	size_t byteRequirement = sizeof(char) * uniqueLen;
	hipMalloc(&flags, byteRequirement); gpuerr();
	hipMalloc(&uniqueDistances, byteRequirement); gpuerr();
	hipMalloc(&distanceOutput, byteRequirement); gpuerr();
	hipMalloc(&pairOutput, sizeof(Int2)*uniqueLen); gpuerr();
	cal_levenshtein <<< NUM_BLOCK(uniqueLen), NUM_THREADS>>>(
	    seq, uniquePairs, distance, uniqueDistances, flags, uniqueLen, seqLen); gpuerr();

	// filter levenshtein
	double_flag(uniquePairs, uniqueDistances, flags, pairOutput,
	            distanceOutput, buffer, uniqueLen); gpuerr();

	print_gpu_memory();
	_cudaFree(uniquePairs, uniqueDistances, flags); gpuerr();
	int outputLen = transfer_last_element(buffer, 1); gpuerr();
	return outputLen;
}

void make_output(Int2* pairOut, char* distanceOut, int len, XTNOutput &output) {
	output.indexPairs = device_to_host(pairOut, len); gpuerr();
	output.pairwiseDistances = device_to_host(distanceOut, len); gpuerr();
	output.len = len;
}

void gen_next_chunk(Chunk<Int3> &keyInOut, Chunk<int> &valueInOut,
                    int* valueOffsets, int offsetLen, int lowerbound, int* buffer) {
	char* flags;
	Int3* keyOut;
	int* valueOut;

	hipMalloc(&flags, sizeof(char)*valueInOut.len); gpuerr();
	hipMemset(flags, 1, sizeof(char)*valueInOut.len); gpuerr();
	hipMalloc(&keyOut, sizeof(Int3)*valueInOut.len); gpuerr();
	hipMalloc(&valueOut, sizeof(int)*valueInOut.len); gpuerr();

	flag_lowerbound <<< NUM_BLOCK(offsetLen), NUM_THREADS>>>(
	    valueInOut.ptr, valueOffsets, flags, lowerbound, offsetLen); gpuerr();
	double_flag(keyInOut.ptr, valueInOut.ptr, flags, keyOut, valueOut,
	            buffer, valueInOut.len); gpuerr();

	int outputLen = transfer_last_element(buffer, 1); gpuerr();
	print_gpu_memory();
	_cudaFree(flags); gpuerr();
	keyInOut.ptr = keyOut;
	keyInOut.len = outputLen;
	valueInOut.ptr = valueOut;
	valueInOut.len = outputLen;
}

int solve_next_bin(int* chunksizes, int start, int maxSize, int n) {
	int ans = 0, len = 0;
	for (int i = start; i < n; i++) {
		int currentChunkSize = chunksizes[i];
		if (len + currentChunkSize > maxSize)
			break;
		len += currentChunkSize;
		ans++;
	}
	return ans;
}

//=====================================
// Public Functions
//=====================================

int solve_bin_packing_lowerbounds(int* histograms, int* &lowerboundsOutput,
                                  int n, int seqLen, int* buffer, MemoryContext ctx) {
	int* rowIndex, *output, *key, *value;
	size_t* histogramIntermediate;

	int nLevel = ctx.histogramSize, len2d = n * nLevel;
	hipMalloc(&rowIndex, sizeof(int) * len2d); gpuerr();
	hipMalloc(&output, sizeof(int) * nLevel); gpuerr();
	hipMalloc(&key, sizeof(int) * nLevel); gpuerr();
	hipMalloc(&value, sizeof(int) * nLevel); gpuerr();
	hipMalloc(&histogramIntermediate, sizeof(size_t) * len2d); gpuerr();

	make_row_index <<< NUM_BLOCK(n), NUM_THREADS>>>(rowIndex, n, nLevel); gpuerr();
	inclusive_sum_by_key(rowIndex, histograms, histogramIntermediate, len2d); gpuerr();
	gen_bounds <<< NUM_BLOCK(nLevel), NUM_THREADS >>>(
	    histogramIntermediate, key, value, 10 /*ctx.maxThroughputExponent*/, seqLen, n, nLevel); gpuerr();
	print_int_arr(key, nLevel);
	max_by_key(key, value, output, buffer, nLevel); gpuerr();

	int outputLen = transfer_last_element(buffer, 1); gpuerr();
	lowerboundsOutput = device_to_host(output, outputLen); gpuerr();

	print_gpu_memory();
	_cudaFree(rowIndex, output, key, value, histogramIntermediate); gpuerr();
	return outputLen;
}

int solve_bin_packing_offsets(int* histograms, int** &offsetOutput,
                              int n, int* buffer, MemoryContext ctx) {
	int* rowIndex, *assignment, *output1d;
	int offsetLen;

	int nLevel = ctx.histogramSize, len2d = n * nLevel;
	hipMalloc(&rowIndex, sizeof(int) * len2d); gpuerr();
	hipMalloc(&assignment, sizeof(int) * len2d); gpuerr();
	hipMalloc(&output1d, sizeof(int) * len2d); gpuerr();
	hipHostMalloc(&offsetOutput, sizeof(int*) * n); gpuerr();

	//solve bin packing
	make_row_index <<< NUM_BLOCK(n), NUM_THREADS>>>(rowIndex, n, nLevel); gpuerr();
	inclusive_sum_by_key(rowIndex, histograms, len2d); gpuerr();
	gen_assignment <<< NUM_BLOCK(nLevel), NUM_THREADS >>>(
	    histograms, assignment, ctx.maxThroughputExponent, n, nLevel); gpuerr();
	max_by_key(assignment, histograms, output1d, buffer, len2d); gpuerr();

	//make output
	int outputLen = transfer_last_element(buffer, 1); gpuerr();

	if (outputLen % n == 0) {
		offsetLen = outputLen / n;
		int* output1dPtr = output1d;
		for (int i = 0; i < n; i++) {
			offsetOutput[i] = device_to_host(output1dPtr, offsetLen); gpuerr();
			output1dPtr += offsetLen;
		}
	} else if (outputLen == 1) {
		for (int i = 0; i < n; i++)
			offsetOutput[i] = device_to_host( histograms + (i * nLevel) + nLevel - 1, 1); gpuerr();
		offsetLen = 1;
	} else
		print_err("bin_packing outputLen is not divisible by inputLen");

	print_gpu_memory();
	_cudaFree(rowIndex, assignment, output1d); gpuerr();
	return offsetLen;
}

void stream_handler1(Chunk<Int3> input, Int3* &deletionsOutput, int* &indexOutput,
                     std::vector<int*> &histogramOutput, int &outputLen, int distance, MemoryContext ctx) {
	int *combinationOffsets;
	unsigned int *histogramValue;
	int* histogram;

	// cal combinationOffsets
	hipMalloc(&combinationOffsets, sizeof(int)*input.len);	gpuerr();
	cal_combination_len <<< NUM_BLOCK(input.len), NUM_THREADS >>>(
	    input.ptr, distance, combinationOffsets, input.len); gpuerr();
	inclusive_sum(combinationOffsets, input.len); gpuerr();
	outputLen = transfer_last_element(combinationOffsets, input.len); gpuerr();

	// generate combinations
	hipMalloc(&deletionsOutput, sizeof(Int3)*outputLen); gpuerr();
	hipMalloc(&indexOutput, sizeof(int)*outputLen); gpuerr();
	hipMalloc(&histogramValue, sizeof(unsigned int)*outputLen); gpuerr();
	gen_combination <<< NUM_BLOCK(input.len), NUM_THREADS >>> (
	    input.ptr, combinationOffsets, distance,
	    deletionsOutput, indexOutput, histogramValue, input.len); gpuerr();

	// generate histogram
	sort_key_values(deletionsOutput, indexOutput, outputLen); gpuerr();
	hipMalloc(&histogram, sizeof(int)*ctx.histogramSize);	gpuerr();
	cal_histogram(histogramValue, histogram, ctx.histogramSize, UINT_MIN, UINT_MAX, outputLen); gpuerr();
	histogramOutput.push_back(histogram);

	printf("total allocation 1 %'lu\n",
	       sizeof(int)*input.len + sizeof(Int3)*outputLen + sizeof(int)*outputLen + sizeof(unsigned int)*outputLen);
	print_gpu_memory();
	_cudaFree(combinationOffsets, histogramValue); gpuerr();
}

void stream_handler2(Chunk<Int3> &keyInOut, Chunk<int> &valueInOut, std::vector<int*> &histogramOutput,
                     int distance, int seqLen, int* buffer, MemoryContext ctx) {
	int* inputOffsets, *valueLengths, *indexes, *valueLengthsHost, *histogram;

	sort_key_values(keyInOut.ptr, valueInOut.ptr, keyInOut.len); gpuerr();
	int offsetLen =
	    cal_offsets(keyInOut.ptr, inputOffsets, valueLengths, keyInOut.len, buffer);

	int start = 0, carry = 0, nChunk;
	int* inputOffsetsPtr = inputOffsets, *valueLengthsPtr = valueLengths;
	valueLengthsHost = device_to_host(valueLengths, offsetLen); gpuerr();

	size_t nChunkSum = 0;
	//histogram loop
	while ((nChunk = solve_next_bin(valueLengthsHost, start, ctx.bandwidth2, offsetLen)) > 0) {

		int chunkLen = gen_smaller_index(
		                   valueInOut.ptr, inputOffsetsPtr, valueLengthsPtr, indexes, carry, nChunk);
		print_bandwidth(chunkLen, ctx.bandwidth2, "2b");
		hipMalloc(&histogram, sizeof(int)*ctx.histogramSize);	gpuerr();
		cal_histogram(indexes, histogram, ctx.histogramSize , 0, seqLen, chunkLen); gpuerr();
		histogramOutput.push_back(histogram);

		carry = transfer_last_element(inputOffsetsPtr, nChunk);
		start += nChunk;
		inputOffsetsPtr += nChunk;
		valueLengthsPtr += nChunk;
		nChunkSum += nChunk;
		print_gpu_memory();
		hipFree(indexes); gpuerr();
	}

	_cudaFree(inputOffsets, valueLengths); gpuerr();
	hipHostFree(valueLengthsHost); gpuerr();
}

void stream_handler3(Chunk<Int3> &keyInOut, Chunk<int> &valueInOut, void callback(Int2*, int),
                     std::vector<int*> &histogramOutput, int lowerbound, int seqLen, int* buffer, MemoryContext ctx) {
	int* inputOffsets, *valueLengths, *valueLengthsHost, *lesserIndex, *histogram;
	Int2* pairOutput;

	int offsetLen = cal_offsets_lowerbound(
	                    keyInOut.ptr, valueInOut.ptr, inputOffsets,
	                    valueLengths, lowerbound, keyInOut.len, buffer);

	int start = 0, carry = 0, nChunk;
	int* inputOffsetsPtr = inputOffsets, *valueLengthsPtr = valueLengths;
	valueLengthsHost = device_to_host(valueLengths, offsetLen); gpuerr();

	// generate pairs
	while ((nChunk = solve_next_bin(valueLengthsHost, start, ctx.bandwidth2, offsetLen)) > 0) {
		int chunkLen = gen_pairs(valueInOut.ptr, inputOffsetsPtr, valueLengthsPtr,
		                         pairOutput, lesserIndex, lowerbound, carry, nChunk);
		print_bandwidth(chunkLen, ctx.bandwidth2, "3b");
		callback(pairOutput, chunkLen);
		hipMalloc(&histogram, sizeof(int)*ctx.histogramSize);	gpuerr();
		cal_histogram(lesserIndex, histogram, ctx.histogramSize , 0, seqLen, chunkLen); gpuerr();
		histogramOutput.push_back(histogram);

		carry = transfer_last_element(inputOffsetsPtr, nChunk);
		start += nChunk;
		inputOffsetsPtr += nChunk;
		valueLengthsPtr += nChunk;

		print_gpu_memory();
		_cudaFree(pairOutput, lesserIndex); gpuerr();
	}

	gen_next_chunk(keyInOut, valueInOut, inputOffsets, offsetLen, lowerbound, buffer);

	_cudaFree(inputOffsets, valueLengths); gpuerr();
	hipHostFree(valueLengthsHost); gpuerr();
}

void stream_handler4(Chunk<Int2> pairInput, XTNOutput & output, Int3 * seq1,
                     int seq1Len, int distance, int* buffer) {
	Int2* pairOut;
	char* distanceOut;
	int outputLen =
	    postprocessing(seq1, pairInput.ptr, distance, pairOut, distanceOut,
	                   pairInput.len, buffer, seq1Len);

	make_output(pairOut, distanceOut, outputLen, output);
	_cudaFree(pairOut, distanceOut); gpuerr();
}