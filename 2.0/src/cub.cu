#include <cub/device/device_scan.cuh>
#include <cub/device/device_merge_sort.cuh>
#include <cub/device/device_run_length_encode.cuh>
#include <cub/device/device_select.cuh>
#include <cub/device/device_histogram.cuh>
#include <cub/device/device_reduce.cuh>
#include "codec.cu"

/**
 * @file
 * Listing of all GPU parallel primitives that use CUB library
 * (everything other than map and expand). Follows Facade design pattern.
 */

struct Int3Comparator {
	HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
	bool operator()(const Int3 &lhs, const Int3 &rhs) {
		/*intentionally sort the second int first as it makes histograms more evenly distributed*/
		if (lhs.entry[1] != rhs.entry[1])
			return lhs.entry[1] < rhs.entry[1];
		if (lhs.entry[0] != rhs.entry[0])
			return lhs.entry[0] < rhs.entry[0];
		// make sure it's irreflexive https://en.cppreference.com/w/cpp/concepts/strict_weak_order
		if (lhs.entry[2] == rhs.entry[2])
			return false;
		return lhs.entry[2] < rhs.entry[2];
	}
};

struct Int2Comparator {
	HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
	bool operator()(const Int2 &lhs, const Int2 &rhs) {
		if (lhs.x != rhs.x)
			return lhs.x < rhs.x;
		// make sure it's irreflexive https://en.cppreference.com/w/cpp/concepts/strict_weak_order
		if (lhs.y == rhs.y)
			return false;
		return lhs.y < rhs.y;
	}
};

struct IntMax {
	HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
	int operator()(const int &a, const int &b) const {
		return (b > a) ? b : a;
	}
};

struct Sum {
	HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
	int operator()(const size_t &a, const size_t &b) const {
		return a + b;
	}
};

template <typename T>
void inclusive_sum(T* input, T* output, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	hipcub::DeviceScan::InclusiveSum(buffer, bufferSize, input, output, n); gpuerr();
	hipMalloc(&buffer, bufferSize); gpuerr(); /*<1% memory*/
	hipcub::DeviceScan::InclusiveSum(buffer, bufferSize, input, output, n); gpuerr();
	hipFree(buffer); gpuerr();
}

template <typename T>
void inclusive_sum(T* input, int n) {
	inclusive_sum(input, input, n);
}

void sort_key_values(Int3* keys, int* values, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	Int3Comparator op;
	hipcub::DeviceMergeSort::SortPairs(buffer, bufferSize, keys, values, n, op); gpuerr();
	hipMalloc(&buffer, bufferSize); gpuerr(); /*16x memory*/
	hipcub::DeviceMergeSort::SortPairs(buffer, bufferSize, keys, values, n, op); gpuerr();
	hipFree(buffer); gpuerr();
}

void sort_int2(Int2* input, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	Int2Comparator op;
	hipcub::DeviceMergeSort::SortKeys(buffer, bufferSize, input, n, op); gpuerr();
	hipMalloc(&buffer, bufferSize); gpuerr(); /*8x memory*/
	hipcub::DeviceMergeSort::SortKeys(buffer, bufferSize, input, n, op); gpuerr();
	hipFree(buffer); gpuerr();
}

template <typename T>
void unique_counts(T* keys, int* output, int* outputLen, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	T* dummy;
	hipMalloc(&dummy, sizeof(T)*n); gpuerr();
	hipcub::DeviceRunLengthEncode::Encode(
	    buffer, bufferSize, keys, dummy, output, outputLen, n); gpuerr();
	hipMalloc(&buffer, bufferSize); gpuerr(); /*~5% memory*/
	hipcub::DeviceRunLengthEncode::Encode(
	    buffer, bufferSize, keys, dummy, output, outputLen, n); gpuerr();
	hipFree(buffer); gpuerr();
	hipFree(dummy); gpuerr();
}

void unique(Int2* input, Int2* output, int* outputLen, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	hipcub::DeviceSelect::Unique(buffer, bufferSize, input, output, outputLen, n); gpuerr();
	hipMalloc(&buffer, bufferSize); gpuerr(); /*~1% memory*/
	hipcub::DeviceSelect::Unique(buffer, bufferSize, input, output, outputLen, n); gpuerr();
	hipFree(buffer); gpuerr();
}

template <typename T1>
void flag(T1* input1, char* flags, T1* output1, int* outputLen, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0, bufferSize2 = 0;
	hipcub::DeviceSelect::Flagged(buffer, bufferSize, input1, flags, output1, outputLen, n); gpuerr();
	hipMalloc(&buffer, bufferSize); gpuerr(); /*~2% memory*/
	hipcub::DeviceSelect::Flagged(buffer, bufferSize, input1, flags, output1, outputLen, n); gpuerr();
	hipFree(buffer); gpuerr();
}

template <typename T>
void cal_histogram(T* input, int* output, int nLevel, T minValue, T maxValue, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	hipcub::DeviceHistogram::HistogramEven(buffer, bufferSize,
	                                    input, output, nLevel + 1, minValue, maxValue, n); gpuerr();
	hipMalloc(&buffer, bufferSize); gpuerr(); /*5-10% memory*/
	hipcub::DeviceHistogram::HistogramEven(buffer, bufferSize,
	                                    input, output, nLevel + 1, minValue, maxValue, n); gpuerr();
	hipFree(buffer); gpuerr();
}

template <typename T>
void inclusive_sum_by_key(int* keyIn, T* valueInOut, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	hipcub::DeviceScan::InclusiveSumByKey(
	    buffer, bufferSize, keyIn, valueInOut, valueInOut, n); gpuerr();
	hipMalloc(&buffer, bufferSize); gpuerr(); /*2% memory*/
	hipcub::DeviceScan::InclusiveSumByKey(
	    buffer, bufferSize, keyIn, valueInOut, valueInOut, n); gpuerr();
	hipFree(buffer); gpuerr();
}

void max_by_key(int* keyIn, int* valueIn, int* valueOut, int* outputLen, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	int* dummy;
	IntMax op;

	hipMalloc(&dummy, sizeof(int)*n); gpuerr();
	hipcub::DeviceReduce::ReduceByKey(buffer, bufferSize, keyIn,
	                               dummy, valueIn, valueOut, outputLen, op, n); gpuerr();
	hipMalloc(&buffer, bufferSize); gpuerr(); /*3% memory*/
	hipcub::DeviceReduce::ReduceByKey(buffer, bufferSize, keyIn,
	                               dummy, valueIn, valueOut, outputLen, op, n); gpuerr();
	_cudaFree(buffer, dummy); gpuerr();
}

void sum_by_key(Int2* keyIn, Int2* keyOut, size_t* valueIn, size_t* valueOut, int* outputLen, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	Sum op;

	hipcub::DeviceReduce::ReduceByKey(buffer, bufferSize, keyIn,
	                               keyOut, valueIn, valueOut, outputLen, op, n); gpuerr();
	hipMalloc(&buffer, bufferSize); gpuerr();
	hipcub::DeviceReduce::ReduceByKey(buffer, bufferSize, keyIn,
	                               keyOut, valueIn, valueOut, outputLen, op, n); gpuerr();
	_cudaFree(buffer); gpuerr();
}