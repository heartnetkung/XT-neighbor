#include <cub/device/device_scan.cuh>
#include <cub/device/device_merge_sort.cuh>
#include <cub/device/device_run_length_encode.cuh>
#include <cub/device/device_select.cuh>
#include <cub/device/device_histogram.cuh>
#include <cub/device/device_reduce.cuh>
#include "codec.cu"

/**
 * @file
 * @brief Listing of all GPU parallel primitives that use CUB library
 * (everything other than map and expand). Follows Facade design pattern.
 */

struct Int3Comparator {
	HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
	bool operator()(const Int3 &lhs, const Int3 &rhs) {
		/*intentionally sort the second int first as it makes histograms more evenly distributed*/
		if (lhs.entry[1] != rhs.entry[1])
			return lhs.entry[1] < rhs.entry[1];
		if (lhs.entry[0] != rhs.entry[0])
			return lhs.entry[0] < rhs.entry[0];
		// make sure it's irreflexive https://en.cppreference.com/w/cpp/concepts/strict_weak_order
		if (lhs.entry[2] == rhs.entry[2])
			return false;
		return lhs.entry[2] < rhs.entry[2];
	}
};

struct Int2Comparator {
	HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
	bool operator()(const Int2 &lhs, const Int2 &rhs) {
		if (lhs.x != rhs.x)
			return lhs.x < rhs.x;
		// make sure it's irreflexive https://en.cppreference.com/w/cpp/concepts/strict_weak_order
		if (lhs.y == rhs.y)
			return false;
		return lhs.y < rhs.y;
	}
};

struct IntMax {
	HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
	int operator()(const int &a, const int &b) const {
		return (b > a) ? b : a;
	}
};

template <typename T1, typename T2>
void inclusive_sum(T1* input, T2* output, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	hipcub::DeviceScan::InclusiveSum(buffer, bufferSize, input, output, n);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceScan::InclusiveSum(buffer, bufferSize, input, output, n);
	hipFree(buffer);
}

template <typename T>
void inclusive_sum(T* input, int n) {
	inclusive_sum(input, input, n);
}

void sort_key_values(Int3* keys, int* values, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	Int3Comparator op;
	hipcub::DeviceMergeSort::SortPairs(buffer, bufferSize, keys, values, n, op);gpuerr();
	hipMalloc(&buffer, bufferSize);gpuerr();
	hipcub::DeviceMergeSort::SortPairs(buffer, bufferSize, keys, values, n, op);gpuerr();
	hipFree(buffer);gpuerr();
}

void sort_int2(Int2* input, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	Int2Comparator op;
	hipcub::DeviceMergeSort::SortKeys(buffer, bufferSize, input, n, op);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceMergeSort::SortKeys(buffer, bufferSize, input, n, op);
	hipFree(buffer);
}

template <typename T>
void unique_counts(T* keys, int* output, int* outputLen, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	T* dummy;
	hipMalloc(&dummy, sizeof(T)*n);
	hipcub::DeviceRunLengthEncode::Encode(
	    buffer, bufferSize, keys, dummy, output, outputLen, n);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceRunLengthEncode::Encode(
	    buffer, bufferSize, keys, dummy, output, outputLen, n);
	hipFree(buffer);
	hipFree(dummy);
}

void unique(Int2* input, Int2* output, int* outputLen, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	hipcub::DeviceSelect::Unique(buffer, bufferSize, input, output, outputLen, n);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceSelect::Unique(buffer, bufferSize, input, output, outputLen, n);
	hipFree(buffer);
}

template <typename T1, typename T2>
void double_flag(T1* input1, T2* input2, char* flags, T1* output1, T2* output2, int* outputLen, int n) {
	void *buffer = NULL, *buffer2 = NULL;
	size_t bufferSize = 0, bufferSize2 = 0;
	hipcub::DeviceSelect::Flagged(buffer, bufferSize, input1, flags, output1, outputLen, n);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceSelect::Flagged(buffer, bufferSize, input1, flags, output1, outputLen, n);

	hipcub::DeviceSelect::Flagged(buffer2, bufferSize2, input2, flags, output2, outputLen, n);
	hipMalloc(&buffer2, bufferSize2);
	hipcub::DeviceSelect::Flagged(buffer2, bufferSize2, input2, flags, output2, outputLen, n);
	_cudaFree(buffer, buffer2);
}

template <typename T>
void cal_histogram(T* input, int* output, int nLevel, T minValue, T maxValue, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	hipcub::DeviceHistogram::HistogramEven(buffer, bufferSize,
	                                    input, output, nLevel + 1, minValue, maxValue, n); gpuerr();
	hipMalloc(&buffer, bufferSize); gpuerr();
	hipcub::DeviceHistogram::HistogramEven(buffer, bufferSize,
	                                    input, output, nLevel + 1, minValue, maxValue, n); gpuerr();
	hipFree(buffer); gpuerr();
}

template <typename T>
void inclusive_sum_by_key(int* keyIn, int* valueIn, T* valueOut, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	hipcub::DeviceScan::InclusiveSumByKey(
	    buffer, bufferSize, keyIn, valueIn, valueOut, n);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceScan::InclusiveSumByKey(
	    buffer, bufferSize, keyIn, valueIn, valueOut, n);
	hipFree(buffer);
}

void inclusive_sum_by_key(int* keyIn, int* valueInOut, int n) {
	inclusive_sum_by_key(keyIn, valueInOut, valueInOut, n);
}

void max_by_key(int* keyIn, int* valueIn, int* valueOut, int* outputLen, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	int* dummy;
	IntMax op;

	hipMalloc(&dummy, sizeof(int)*n);
	hipcub::DeviceReduce::ReduceByKey(buffer, bufferSize, keyIn,
	                               dummy, valueIn, valueOut, outputLen, op, n);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceReduce::ReduceByKey(buffer, bufferSize, keyIn,
	                               dummy, valueIn, valueOut, outputLen, op, n);
	_cudaFree(buffer, dummy);
}
