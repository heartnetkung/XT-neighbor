#include <cub/device/device_scan.cuh>
#include <cub/device/device_merge_sort.cuh>
#include <cub/device/device_run_length_encode.cuh>
#include <cub/device/device_select.cuh>
#include <cub/device/device_histogram.cuh>
#include <cub/device/device_reduce.cuh>
#include "codec.cu"

struct Int3Comparator {
	HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
	bool operator()(const Int3 &lhs, const Int3 &rhs) {
		if (lhs.entry[0] != rhs.entry[0])
			return lhs.entry[0] < rhs.entry[0];
		if (lhs.entry[1] != rhs.entry[1])
			return lhs.entry[1] < rhs.entry[1];
		return lhs.entry[2] < rhs.entry[2];
	}
};

struct Int2Comparator {
	HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
	bool operator()(const Int2 &lhs, const Int2 &rhs) {
		if (lhs.x != rhs.x)
			return lhs.x < rhs.x;
		return lhs.y < rhs.y;
	}
};

struct SizeTMax {
	HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
	size_t operator()(const size_t &a, const size_t &b) const {
		return (b > a) ? b : a;
	}
};

template <typename T>
void inclusive_sum(T* input, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	hipcub::DeviceScan::InclusiveSum(buffer, bufferSize, input, input, n);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceScan::InclusiveSum(buffer, bufferSize, input, input, n);
	hipFree(buffer);
}

void inclusive_sum(int* input, size_t* output, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	hipcub::DeviceScan::InclusiveSum(buffer, bufferSize, input, output, n);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceScan::InclusiveSum(buffer, bufferSize, input, output, n);
	hipFree(buffer);
}

void sort_key_values(Int3* keys, int* values, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	Int3Comparator op;
	hipcub::DeviceMergeSort::SortPairs(buffer, bufferSize, keys, values, n, op);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceMergeSort::SortPairs(buffer, bufferSize, keys, values, n, op);
	hipFree(buffer);
}

void sort_int2(Int2* input, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	Int2Comparator op;
	hipcub::DeviceMergeSort::SortKeys(buffer, bufferSize, input, n, op);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceMergeSort::SortKeys(buffer, bufferSize, input, n, op);
	hipFree(buffer);
}

template <typename T>
void unique_counts(T* keys, int* output, int* outputLen, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	T* dummy;
	hipMalloc(&dummy, sizeof(T)*n);
	hipcub::DeviceRunLengthEncode::Encode(
	    buffer, bufferSize, keys, dummy, output, outputLen, n);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceRunLengthEncode::Encode(
	    buffer, bufferSize, keys, dummy, output, outputLen, n);
	hipFree(buffer);
	hipFree(dummy);
}

void unique(Int2* input, Int2* output, int* outputLen, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	hipcub::DeviceSelect::Unique(buffer, bufferSize, input, output, outputLen, n);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceSelect::Unique(buffer, bufferSize, input, output, outputLen, n);
	hipFree(buffer);
}

template <typename T1, typename T2>
void double_flag(T1* input1, T2* input2, char* flags, T1* output1, T2* output2, int* outputLen, int n) {
	void *buffer = NULL, *buffer2 = NULL;
	size_t bufferSize = 0, bufferSize2 = 0;
	hipcub::DeviceSelect::Flagged(buffer, bufferSize, input1, flags, output1, outputLen, n);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceSelect::Flagged(buffer, bufferSize, input1, flags, output1, outputLen, n);
	hipcub::DeviceSelect::Flagged(buffer2, bufferSize2, input2, flags, output2, outputLen, n);
	hipMalloc(&buffer2, bufferSize2);
	hipcub::DeviceSelect::Flagged(buffer2, bufferSize2, input2, flags, output2, outputLen, n);
	_cudaFree(buffer, buffer2);
}

template <typename T>
void histogram(T* input, int* output, int nLevel, T maxValue, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	unsigned int minValue = 0;
	hipcub::DeviceHistogram::HistogramEven(buffer, bufferSize,
	                                    input, output, nLevel + 1, minValue, maxValue, n);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceHistogram::HistogramEven(buffer, bufferSize,
	                                    input, output, nLevel + 1, minValue, maxValue, n);
	hipFree(buffer);
}

void inclusive_sum_by_key(int* keyIn, int* valueIn, size_t* valueOut, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	hipcub::DeviceScan::InclusiveSumByKey(
	    buffer, bufferSize, keyIn, valueIn, valueOut, n);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceScan::InclusiveSumByKey(
	    buffer, bufferSize, keyIn, valueIn, valueOut, n);
	hipFree(buffer);
}

void max_by_key(int* keyIn, size_t* valueIn, size_t* valueOut, int* outputLen, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	int* dummy;
	SizeTMax op;

	printf("1.1\n");
	hipMalloc(&dummy, sizeof(int)*n);
	printf("1.2\n");
	hipcub::DeviceReduce::ReduceByKey(buffer, bufferSize, keyIn,
	                               dummy, valueIn, valueOut, outputLen, op, n);
	printf("1.3\n");
	hipMalloc(&buffer, bufferSize);
	printf("1.4\n");
	hipcub::DeviceReduce::ReduceByKey(buffer, bufferSize, keyIn,
	                               dummy, valueIn, valueOut, outputLen, op, n);
	printf("1.5\n");
	_cudaFree(buffer, dummy);
}
