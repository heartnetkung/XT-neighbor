#include "hip/hip_runtime.h"
#include "xtn.h"
#include <sys/sysinfo.h>
#include <time.h>

int print_err(const char* str) {
	fprintf(stderr, "Error: %s\n", str);
	return ERROR;
}

float startTime = 0;

int clock_start() {
	startTime = (float)clock() / CLOCKS_PER_SEC;
}

void print_args(XTNArgs args) {
	printf("XTNArgs{\n");
	printf("\tdistance: %d\n", args.distance);
	printf("\tverbose: %d\n", args.verbose);
	printf("\tseq1Len: %'d\n", args.seq1Len);
	printf("\tseq1Path: \"%s\"\n", args.seq1Path);
	printf("\toutputPath: \"%s\"\n", args.outputPath);
	printf("}\n");
}

void _cudaFree(void* a) {
	hipFree(a);
}
void _cudaFree(void* a, void* b) {
	hipFree(a);
	hipFree(b);
}
void _cudaFree(void* a, void* b, void* c) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
}
void _cudaFree(void* a, void* b, void* c, void* d) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
	hipFree(e);
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e, void* f) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
	hipFree(e);
	hipFree(f);
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e, void* f, void* g) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
	hipFree(e);
	hipFree(g);
}

void _cudaFreeHost(void* a, void* b) {
	hipHostFree(a);
	hipHostFree(b);
}

void _cudaFreeHost(void* a, void* b, void* c) {
	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}

void _cudaFreeHost(void* a, void* b, void* c, void* d) {
	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
	hipHostFree(d);
}

template <typename T>
void _cudaFreeHost2D(T** a, int n) {
	for (int i = 0; i < n; i++)
		hipHostFree(a[i]);
	hipHostFree(a);
}

void _free(void* a, void* b) {
	free(a);
	free(b);
}

void _free(void* a, void* b, void* c) {
	free(a);
	free(b);
	free(c);
}

int divide_ceil(int a, int b) {
	return (a + b - 1) / b;
}

template <typename T>
T* device_to_host(T* arr, int n) {
	T* temp;
	size_t tempBytes = sizeof(T) * n;
	hipHostMalloc(&temp, tempBytes);
	hipMemcpy(temp, arr, tempBytes, hipMemcpyDeviceToHost);
	return temp;
}

template <typename T>
T* host_to_device(T* arr, int n) {
	T* temp;
	size_t tempBytes = sizeof(T) * n;
	hipMalloc(&temp, tempBytes);
	hipMemcpy(temp, arr, tempBytes, hipMemcpyHostToDevice);
	return temp;
}

#define gpuerr() { print_cuda_error( __FILE__, __LINE__); }

void print_cuda_error(const char *file, int line) {
	hipError_t code = hipGetLastError();
	if (code != hipSuccess)
		printf("Cuda error at %s %s %d\n", hipGetErrorName(code), file, line);
}

void print_int_arr(int* arr, int n) {
	printf("[ ");
	int* arr2 = device_to_host(arr, n);
	for (int i = 0; i < n; i++) {
		printf("%d", arr2[i]);
		if (i != n - 1)
			printf(", ");
	}
	printf(" ] n=%d\n", n);
	if (n > 0)
		hipHostFree(arr2);
}

void print_char_arr(char* arr, int n) {
	printf("[ ");
	char* arr2 = device_to_host(arr, n);
	for (int i = 0; i < n; i++) {
		printf("%d", arr2[i]);
		if (i != n - 1)
			printf(", ");
	}
	printf(" ] n=%d\n", n);
	if (n > 0)
		hipHostFree(arr2);
}

void print_int2_arr(Int2* arr, int n) {
	printf("[ ");
	Int2* arr2 = device_to_host(arr, n);
	for (int i = 0; i < n; i++) {
		printf("(%d %d)", arr2[i].x, arr2[i].y);
		if (i != n - 1)
			printf(", ");
	}
	printf(" ] n=%d\n", n);
	if (n > 0)
		hipHostFree(arr2);
}

void print_size_t_arr(size_t* arr, int n) {
	printf("[ ");
	size_t* arr2 = device_to_host(arr, n);
	for (int i = 0; i < n; i++) {
		printf("%lu", arr2[i]);
		if (i != n - 1)
			printf(", ");
	}
	printf(" ] n=%d\n", n);
	hipHostFree(arr2);
}

void print_gpu_memory() {
	size_t mf, ma;
	hipMemGetInfo(&mf, &ma);
	printf("GPU Memory: %'lu / %'lu\n", mf, ma);
}

void print_main_memory() {
	struct sysinfo si;
	sysinfo (&si);
	printf("Main Memory: %'lu / %'lu\n", si.freeram, si.totalram);
}

void print_tp(int verbose, const char* step, size_t throughput) {
	if (verbose)
		printf("step %s completed with throughput: %'lu\n", step, throughput);
}

void print_bandwidth(int chunkLen, int bandwidth, const char* process) {
	float endTime = (float)clock() / CLOCKS_PER_SEC;
	printf("process %s started with bandwidth %'d / %'d %'.0f\n",
	       process, chunkLen, bandwidth, endTime - startTime);
}

void print_v(int verbose, const char* message) {
	if (verbose)
		printf("%s\n", message);
}

size_t get_gpu_memory() {
	size_t mf, ma;
	hipMemGetInfo(&mf, &ma);
	return mf;
}

size_t get_main_memory() {
	struct sysinfo si;
	sysinfo (&si);
	return si.freeram;
}