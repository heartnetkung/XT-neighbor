#include "hip/hip_runtime.h"
#include "xtn.h"
#include <sys/sysinfo.h>
#include <time.h>

#define gpuerr() { print_cuda_error( __FILE__, __LINE__); }

void print_cuda_error(const char *file, int line) {
	hipError_t code = hipGetLastError();
	if (code != hipSuccess)
		printf("Cuda error at %s %s %d\n", hipGetErrorName(code), file, line);
}

int print_err(const char* str) {
	fprintf(stderr, "Error: %s\n", str);
	return ERROR;
}

float startTime = 0;

void clock_start() {
	startTime = (float)clock() / CLOCKS_PER_SEC;
}

void print_args(XTNArgs args) {
	printf("XTNArgs{\n");
	printf("\tdistance: %d\n", args.distance);
	printf("\tverbose: %d\n", args.verbose);
	printf("\tseq1Len: %'d\n", args.seq1Len);
	printf("\tseq1Path: \"%s\"\n", args.seq1Path);
	printf("\toutputPath: \"%s\"\n", args.outputPath);
	printf("}\n");
}

void _cudaFree(void* a) {
	hipFree(a); gpuerr();
}
void _cudaFree(void* a, void* b) {
	hipFree(a); gpuerr();
	hipFree(b); gpuerr();
}
void _cudaFree(void* a, void* b, void* c) {
	hipFree(a); gpuerr();
	hipFree(b); gpuerr();
	hipFree(c); gpuerr();
}
void _cudaFree(void* a, void* b, void* c, void* d) {
	hipFree(a); gpuerr();
	hipFree(b); gpuerr();
	hipFree(c); gpuerr();
	hipFree(d); gpuerr();
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e) {
	hipFree(a); gpuerr();
	hipFree(b); gpuerr();
	hipFree(c); gpuerr();
	hipFree(d); gpuerr();
	hipFree(e); gpuerr();
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e, void* f) {
	hipFree(a); gpuerr();
	hipFree(b); gpuerr();
	hipFree(c); gpuerr();
	hipFree(d); gpuerr();
	hipFree(e); gpuerr();
	hipFree(f); gpuerr();
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e, void* f, void* g) {
	hipFree(a); gpuerr();
	hipFree(b); gpuerr();
	hipFree(c); gpuerr();
	hipFree(d); gpuerr();
	hipFree(e); gpuerr();
	hipFree(g); gpuerr();
}

void _cudaFreeHost(void* a, void* b) {
	hipHostFree(a); gpuerr();
	hipHostFree(b); gpuerr();
}

void _cudaFreeHost(void* a, void* b, void* c) {
	hipHostFree(a); gpuerr();
	hipHostFree(b); gpuerr();
	hipHostFree(c); gpuerr();
}

void _cudaFreeHost(void* a, void* b, void* c, void* d) {
	hipHostFree(a); gpuerr();
	hipHostFree(b); gpuerr();
	hipHostFree(c); gpuerr();
	hipHostFree(d); gpuerr();
}

template <typename T>
void _cudaFreeHost2D(T** a, int n) {
	for (int i = 0; i < n; i++) {
		hipHostFree(a[i]); gpuerr();
	}
	hipHostFree(a); gpuerr();
}

void _free(void* a, void* b) {
	free(a);
	free(b);
}

void _free(void* a, void* b, void* c) {
	free(a);
	free(b);
	free(c);
}

int divide_ceil(int a, int b) {
	return (a + b - 1) / b;
}

template <typename T>
T* device_to_host(T* arr, int n) {
	T* temp;
	size_t tempBytes = sizeof(T) * n;
	hipHostMalloc(&temp, tempBytes); gpuerr();
	hipMemcpy(temp, arr, tempBytes, hipMemcpyDeviceToHost); gpuerr();
	return temp;
}

template <typename T>
T* host_to_device(T* arr, int n) {
	T* temp;
	size_t tempBytes = sizeof(T) * n;
	hipMalloc(&temp, tempBytes); gpuerr();
	hipMemcpy(temp, arr, tempBytes, hipMemcpyHostToDevice); gpuerr();
	return temp;
}

void print_int_arr(int* arr, int n) {
	printf("[ ");
	int* arr2 = device_to_host(arr, n);
	for (int i = 0; i < n; i++) {
		printf("%d", arr2[i]);
		if (i != n - 1)
			printf(", ");
	}
	printf(" ] n=%d\n", n);
	if (n > 0) {
		hipHostFree(arr2); gpuerr();
	}
}

void print_char_arr(char* arr, int n) {
	printf("[ ");
	char* arr2 = device_to_host(arr, n);
	for (int i = 0; i < n; i++) {
		printf("%d", arr2[i]);
		if (i != n - 1)
			printf(", ");
	}
	printf(" ] n=%d\n", n);
	if (n > 0) {
		hipHostFree(arr2); gpuerr();
	}
}

void print_int2_arr(Int2* arr, int n) {
	printf("[ ");
	Int2* arr2 = device_to_host(arr, n);
	for (int i = 0; i < n; i++) {
		printf("(%d %d)", arr2[i].x, arr2[i].y);
		if (i != n - 1)
			printf(", ");
	}
	printf(" ] n=%d\n", n);
	if (n > 0) {
		hipHostFree(arr2); gpuerr();
	}
}

void print_size_t_arr(size_t* arr, int n) {
	printf("[ ");
	size_t* arr2 = device_to_host(arr, n);
	for (int i = 0; i < n; i++) {
		printf("%lu", arr2[i]);
		if (i != n - 1)
			printf(", ");
	}
	printf(" ] n=%d\n", n);
	hipHostFree(arr2); gpuerr();
}

void print_gpu_memory() {
	size_t mf, ma;
	hipMemGetInfo(&mf, &ma);
	printf("GPU Memory: %'lu / %'lu\n", mf, ma);
}

void print_main_memory() {
	struct sysinfo si;
	sysinfo (&si);
	printf("Main Memory: %'lu / %'lu\n", si.freeram, si.totalram);
}

void print_tp(int verbose, const char* step, size_t throughput) {
	if (verbose)
		printf("step %s completed with throughput: %'lu\n", step, throughput);
}

void print_bandwidth(int chunkLen, int bandwidth, const char* process) {
	float endTime = (float)clock() / CLOCKS_PER_SEC;
	printf("process %s started with bandwidth %'d / %'d %'.0f\n",
	       process, chunkLen, bandwidth, endTime - startTime);
}

void print_v(int verbose, const char* message) {
	if (verbose)
		printf("%s\n", message);
}

size_t get_gpu_memory() {
	size_t mf, ma;
	hipMemGetInfo(&mf, &ma);
	return mf;
}

size_t get_main_memory() {
	struct sysinfo si;
	sysinfo (&si);
	return si.freeram;
}