#include "hip/hip_runtime.h"
#include "xtn.h"
#include <sys/sysinfo.h>
#include <time.h>

#define gpuerr() { print_cuda_error( __FILE__, __LINE__); }

void print_cuda_error(const char *file, int line) {
	hipError_t code = hipGetLastError();
	if (code != hipSuccess)
		printf("Cuda error at %s %s %d\n", hipGetErrorName(code), file, line);
}

int print_err(const char* str) {
	fprintf(stderr, "Error: %s\n", str);
	return ERROR;
}

int print_err_line(const char* str, int line) {
	fprintf(stderr, "Error: %s\n", str);
	fprintf(stderr, "Error line: %d\n", line);
	return ERROR;
}

float startTime = 0;

float clock_start() {
	if (verboseGlobal) {
		startTime = (float)clock() / CLOCKS_PER_SEC;
		return startTime;
	}
	return 0;
}

float get_time() {
	if (verboseGlobal)
		return (float)clock() / CLOCKS_PER_SEC;
	return 0;
}

void print_args(XTNArgs args) {
	printf("XTNArgs{\n");
	printf("\tdistance: %d\n", args.distance);
	printf("\tverbose: %d\n", args.verbose);
	printf("\tseqLen: %'d\n", args.seqLen);
	printf("\tseqPath: \"%s\"\n", args.seqPath);
	printf("\toutputPath: \"%s\"\n", args.outputPath);
	printf("\tmeasure: \"%s\"\n", (args.measure == LEVENSHTEIN) ? "leven" : "hamming");
	printf("\tinfoPath: \"%s\"\n", args.infoPath);
	printf("\tinfoLen: %'d\n", args.infoLen);
	printf("}\n");
}

template <typename T1, typename T2>
void _cudaMalloc(T1* &a, T2* &b, size_t len) {
	hipMalloc(&a, sizeof(T1)*len); gpuerr();
	hipMalloc(&b, sizeof(T2)*len); gpuerr();
}
template <typename T1, typename T2, typename T3>
void _cudaMalloc(T1* &a, T2* &b, T3* &c, size_t len) {
	hipMalloc(&a, sizeof(T1)*len); gpuerr();
	hipMalloc(&b, sizeof(T2)*len); gpuerr();
	hipMalloc(&c, sizeof(T3)*len); gpuerr();
}
void _cudaFree(void* a) {
	hipFree(a); gpuerr();
}
void _cudaFree(void* a, void* b) {
	hipFree(a); gpuerr();
	hipFree(b); gpuerr();
}
void _cudaFree(void* a, void* b, void* c) {
	hipFree(a); gpuerr();
	hipFree(b); gpuerr();
	hipFree(c); gpuerr();
}
void _cudaFree(void* a, void* b, void* c, void* d) {
	hipFree(a); gpuerr();
	hipFree(b); gpuerr();
	hipFree(c); gpuerr();
	hipFree(d); gpuerr();
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e) {
	hipFree(a); gpuerr();
	hipFree(b); gpuerr();
	hipFree(c); gpuerr();
	hipFree(d); gpuerr();
	hipFree(e); gpuerr();
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e, void* f) {
	hipFree(a); gpuerr();
	hipFree(b); gpuerr();
	hipFree(c); gpuerr();
	hipFree(d); gpuerr();
	hipFree(e); gpuerr();
	hipFree(f); gpuerr();
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e, void* f, void* g) {
	hipFree(a); gpuerr();
	hipFree(b); gpuerr();
	hipFree(c); gpuerr();
	hipFree(d); gpuerr();
	hipFree(e); gpuerr();
	hipFree(g); gpuerr();
}

void _cudaFreeHost(void* a, void* b) {
	hipHostFree(a); gpuerr();
	hipHostFree(b); gpuerr();
}

void _cudaFreeHost(void* a, void* b, void* c) {
	hipHostFree(a); gpuerr();
	hipHostFree(b); gpuerr();
	hipHostFree(c); gpuerr();
}

void _cudaFreeHost(void* a, void* b, void* c, void* d) {
	hipHostFree(a); gpuerr();
	hipHostFree(b); gpuerr();
	hipHostFree(c); gpuerr();
	hipHostFree(d); gpuerr();
}

template <typename T>
void _cudaFreeHost2D(T** a, int n) {
	for (int i = 0; i < n; i++) {
		hipHostFree(a[i]); gpuerr();
	}
	hipHostFree(a); gpuerr();
}

void _free(void* a, void* b) {
	free(a);
	free(b);
}

void _free(void* a, void* b, void* c) {
	free(a);
	free(b);
	free(c);
}

int divide_ceil(int a, int b) {
	return (a + b - 1) / b;
}

template <typename T>
T* device_to_host(T* arr, int n) {
	T* temp;
	size_t tempBytes = sizeof(T) * n;
	hipHostMalloc(&temp, tempBytes); gpuerr();
	hipMemcpy(temp, arr, tempBytes, hipMemcpyDeviceToHost); gpuerr();
	return temp;
}

template <typename T>
T* host_to_device(T* arr, int n) {
	T* temp;
	size_t tempBytes = sizeof(T) * n;
	hipMalloc(&temp, tempBytes); gpuerr();
	hipMemcpy(temp, arr, tempBytes, hipMemcpyHostToDevice); gpuerr();
	return temp;
}

template <typename T>
T* shrink(T* arr, int n) {
	T* temp;
	size_t tempBytes = sizeof(T) * n;
	hipMalloc(&temp, tempBytes); gpuerr();
	hipMemcpy(temp, arr, tempBytes, hipMemcpyDeviceToDevice); gpuerr();
	hipFree(arr); gpuerr();
	return temp;
}

void print_int_arr(int* arr, int n) {
	printf("[ ");
	int* arr2 = device_to_host(arr, n);
	for (int i = 0; i < n; i++) {
		printf("%d", arr2[i]);
		if (i != n - 1)
			printf(", ");
	}
	printf(" ] n=%d\n", n);
	if (n > 0) {
		hipHostFree(arr2); gpuerr();
	}
}

void print_char_arr(char* arr, int n) {
	printf("[ ");
	char* arr2 = device_to_host(arr, n);
	for (int i = 0; i < n; i++) {
		printf("%d", arr2[i]);
		if (i != n - 1)
			printf(", ");
	}
	printf(" ] n=%d\n", n);
	if (n > 0) {
		hipHostFree(arr2); gpuerr();
	}
}

void print_int2_arr(Int2* arr, int n) {
	printf("[ ");
	Int2* arr2 = device_to_host(arr, n);
	for (int i = 0; i < n; i++) {
		printf("(%d %d)", arr2[i].x, arr2[i].y);
		if (i != n - 1)
			printf(", ");
	}
	printf(" ] n=%d\n", n);
	if (n > 0) {
		hipHostFree(arr2); gpuerr();
	}
}

void print_size_t_arr(size_t* arr, int n) {
	printf("[ ");
	size_t* arr2 = device_to_host(arr, n);
	for (int i = 0; i < n; i++) {
		printf("%lu", arr2[i]);
		if (i != n - 1)
			printf(", ");
	}
	printf(" ] n=%d\n", n);
	hipHostFree(arr2); gpuerr();
}

void print_seqinfo_arr(SeqInfo* arr, int n) {
	printf("[ ");
	SeqInfo* arr2 = device_to_host(arr, n);
	for (int i = 0; i < n; i++) {
		printf("(%d %d %d)", arr2[i].frequency, arr2[i].repertoire, arr2[i].originalIndex);
		if (i != n - 1)
			printf(", ");
	}
	printf(" ] n=%d\n", n);
	hipHostFree(arr2); gpuerr();
}

void print_gpu_memory() {
	size_t mf, ma;
	hipMemGetInfo(&mf, &ma);
	printf("GPU Memory: %'lu / %'lu\n", mf, ma);
}

void print_main_memory() {
	struct sysinfo si;
	sysinfo (&si);
	printf("Main Memory: %'lu / %'lu\n", si.freeram, si.totalram);
}

void print_tl(const char* stream, size_t tl) {
	if (verboseGlobal)
		printf("stream %s completed with total length: %'lu\n", stream, tl);
}

void print_bandwidth(int chunkLen, int bandwidth, const char* process) {
	if (!verboseGlobal)
		return;
	printf("process %s started with bandwidth %'d / %'d\n",
	       process, chunkLen, bandwidth);
}

void print_v(const char* message) {
	if (verboseGlobal)
		printf("%s\n", message);
}

size_t get_gpu_memory() {
	size_t mf, ma;
	hipMemGetInfo(&mf, &ma);
	return mf;
}

size_t get_main_memory() {
	struct sysinfo si;
	sysinfo (&si);
	return si.freeram;
}