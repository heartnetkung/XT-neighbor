#include "xtn.h"

int print_err(const char* str) {
#ifndef TEST_ENV
	fprintf(stderr, "Error: %s\n", str);
#endif
	return ERROR;
}

void print_args(XTNArgs args) {
	printf("XTNArgs{\n");
	printf("\tdistance: %d\n", args.distance);
	printf("\tverbose: %d\n", args.verbose);
	printf("\tseq1Len: %d\n", args.seq1Len);
	printf("\tseq1Path: \"%s\"\n", args.seq1Path);
	printf("\toutputPath: \"%s\"\n", args.outputPath);
	printf("}\n");
}

void _cudaFree(void* a) {
	hipFree(a);
}
void _cudaFree(void* a, void* b) {
	hipFree(a);
	hipFree(b);
}
void _cudaFree(void* a, void* b, void* c) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
}
void _cudaFree(void* a, void* b, void* c, void* d) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
	hipFree(e);
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e, void* f) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
	hipFree(e);
	hipFree(f);
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e, void* f, void* g) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
	hipFree(e);
	hipFree(g);
}

void print_tp(int verbose, const char* step, int throughput) {
	if (verbose)
		printf("step %s completed with throughput: %'d\n", step, throughput);
}

void _cudaFreeHost(void* a, void* b) {
	hipHostFree(a);
	hipHostFree(b);
}

void _free(void* a, void* b) {
	free(a);
	free(b);
}

void _free(void* a, void* b, void* c) {
	free(a);
	free(b);
	free(c);
}

int divide_ceil(int a, int b) {
	return (a + b - 1) / b;
}

template <typename T>
T* device_to_host(T* arr, int n) {
	T* temp;
	int tempBytes = sizeof(T) * n;
	hipHostMalloc(&temp, tempBytes);
	hipMemcpy(temp, arr, tempBytes, hipMemcpyDeviceToHost);
	return temp;
}

template <typename T>
T* host_to_device(T* arr, int n) {
	T* temp;
	int tempBytes = sizeof(T) * n;
	hipMalloc(&temp, tempBytes);
	hipMemcpy(temp, arr, tempBytes, hipMemcpyHostToDevice);
	return temp;
}

#define gpuerr() { print_cuda_error( __FILE__, __LINE__); }

void print_cuda_error(const char *file, int line) {
	hipError_t code = hipGetLastError();
	if (code != hipSuccess)
		printf("Cuda error at %s %s %d\n", hipGetErrorName(code), file, line);
}

void print_int_arr(int* arr, int n) {
	printf("[ ");
	int* arr2 = device_to_host(arr, n);
	for (int i = 0; i < n; i++) {
		printf("%d", arr2[i]);
		if (i != n - 1)
			printf(", ");
	}
	printf(" ] n=%d\n", n);
	hipHostFree(arr2);
}

void print_char_arr(char* arr, int n) {
	printf("[ ");
	char* arr2 = device_to_host(arr, n);
	for (int i = 0; i < n; i++) {
		printf("%d", arr2[i]);
		if (i != n - 1)
			printf(", ");
	}
	printf(" ] n=%d\n", n);
	hipHostFree(arr2);
}

void print_int2_arr(Int2* arr, int n) {
	printf("[ ");
	Int2* arr2 = device_to_host(arr, n);
	for (int i = 0; i < n; i++) {
		printf("(%d %d)", arr2[i].x, arr2[i].y);
		if (i != n - 1)
			printf(", ");
	}
	printf(" ] n=%d\n", n);
	hipHostFree(arr2);
}

void print_size_t_arr(size_t* arr, int n) {
	printf("2.2\n");
	printf("[ ");
	size_t* arr2 = device_to_host(arr, n);
	printf("2.3\n");
	for (int i = 0; i < n; i++) {
		printf("2.31\n");
		printf("%lu", arr2[i]);
		printf("2.32\n");
		if (i != n - 1)
			printf(", ");
		printf("2.33\n");
	}
	printf("2.4\n");
	printf(" ] n=%d\n", n);
	printf("2.5\n");
	hipHostFree(arr2);
	printf("2.6\n");
}