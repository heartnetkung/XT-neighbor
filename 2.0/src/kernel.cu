#include "hip/hip_runtime.h"
#include "codec.cu"
#include <limits.h>

const size_t MAX = INT_MAX;

/**
 * @file
 * A collection of most GPU parallel primitives that is implemented as CUDA kernel
 * (most map and expand operations). Follows Facade design pattern.
 */

/**
 * transfer last element of the GPU array to main memory.
 * @param deviceArr the GPU array
 * @param n array length
*/
template <typename T>
T transfer_last_element(T* deviceArr, int n) {
	T ans[1];
	hipMemcpy(ans, deviceArr + n - 1, sizeof(T), hipMemcpyDeviceToHost); gpuerr();
	hipDeviceSynchronize(); gpuerr();
	return ans[0];
}

/**
 * precalculate the number of positions required in the output array of generate combination operation.
 *
 * @param input sequences to generate combination
 * @param distance Levenshtein threshold
 * @param output position output for each sequence
 * @param n array length of input and output
*/
__global__
void cal_combination_len(Int3* input, int distance, int* output, int n) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid >= n)
		return;

	int len = len_decode(input[tid]);
	int newValue = 1 + len;
	if (distance == 2)
		newValue += len * (len - 1) / 2;
	// distance larger than 2 is not supported

	output[tid] = newValue;
}

/**
 * precalculate the number of positions required in the output array of generate pair operation.
 *
 * @param input group size
 * @param output position requirement
 * @param n array length of input and output
*/
__global__
void cal_pair_len(int* input, int* output, int n) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid >= n)
		return;

	size_t intermediate = input[tid];
	intermediate = intermediate * (intermediate - 1) / 2;
	if (intermediate > MAX)
		printf("cal_pair_len overflow\n");
	output[tid] = intermediate;
}

/**
 * precalculate the number of positions required in the output array of generate pair operation with lower bound constratint.
 *
 * @param indexes value of seqIndexes to generate pair
 * @param inputOffsets group offsets
 * @param outputLengths output position requirement
 * @param lowerbound the processing limit for the indexes
 * @param n array length
*/
__global__
void cal_pair_len_lowerbound(int* indexes, int* inputOffsets, int* outputLengths, int lowerbound, int n) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid >= n)
		return;

	int start = tid == 0 ? 0 : inputOffsets[tid - 1];
	int end = inputOffsets[tid];
	int invalidCount = 0;
	for (int i = start; i < end; i++)
		if (indexes[i] > lowerbound)
			invalidCount++;

	size_t intermediate = end - start;
	intermediate = ((intermediate * (intermediate - 1)) - (invalidCount * (invalidCount - 1)) ) / 2;
	if (intermediate > MAX)
		printf("cal_pair_len_lowerbound overflow\n");
	outputLengths[tid] = intermediate;
}

/**
 * combinatorially generate pairs of indexes within the same group.
 *
 * @param indexes value of seqIndexes to generate pair
 * @param outputs pairs output
 * @param inputOffsets precalculated group offsets
 * @param outputOffsets precalculated output position requirement
 * @param lesserIndex by partial output for histogram
 * @param lowerbound the processing limit for the indexes
 * @param carry latest offset from previous chunk in the stream
 * @param n array length of inputOffsets and outputOffsets
*/
__global__
void generate_pairs(int* indexes, Int2* outputs, int* inputOffsets, int* outputOffsets,
                    int* lesserIndex, int lowerbound, int carry, int n) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid >= n)
		return;

	int start = tid == 0 ? carry : inputOffsets[tid - 1];
	int end = inputOffsets[tid];
	int outputIndex = tid == 0 ? 0 : outputOffsets[tid - 1];
	int outputEnd = outputOffsets[tid];

	for (int i = start; i < end; i++) {
		for (int j = i + 1; j < end; j++) {
			Int2 newValue;
			if (indexes[i] < indexes[j]) {
				if (indexes[i] > lowerbound)
					continue;
				newValue.x = indexes[i];
				newValue.y = indexes[j];
			} else {
				if (indexes[j] > lowerbound)
					continue;
				newValue.x = indexes[j];
				newValue.y = indexes[i];
			}
			if (outputIndex < outputEnd) {
				outputs[outputIndex] = newValue;
				lesserIndex[outputIndex++] = newValue.x;
			}
			else
				printf("[1]potential error on generate pairs\n");
		}
	}
}

/**
 * combinatorially generate pairs of indexes within the same group but record only the partial output.
 *
 * @param indexes value of seqIndexes to generate pair
 * @param outputs smaller index output
 * @param inputOffsets precalculated group offsets
 * @param outputOffsets precalculated output position requirement
 * @param carry latest offset from previous chunk in the stream
 * @param n array length of inputOffsets and outputOffsets
*/
__global__
void generate_smaller_index(int* indexes, int* outputs, int* inputOffsets,
                            int* outputOffsets, int carry, int n) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid >= n)
		return;

	int start = tid == 0 ? carry : inputOffsets[tid - 1];
	int end = inputOffsets[tid];
	int outputIndex = tid == 0 ? 0 : outputOffsets[tid - 1];
	int outputEnd = outputOffsets[tid];

	for (int i = start; i < end; i++) {
		for (int j = i + 1; j < end; j++) {
			if (outputIndex < outputEnd)
				outputs[outputIndex++] = indexes[i] < indexes[j] ? indexes[i] : indexes[j];
			else
				printf("[2]potential error on generate pairs\n");
		}
	}
}

#define MIN3(a, b, c) ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)))

/**
 * calculate Levenshtein distance of 2 strings in GPU.
 * @param x1 first string
 * @param x2 second string
*/
__device__
char levenshtein(Int3 x1, Int3 x2) {
	char s1len = (char)len_decode(x1), s2len = (char)len_decode(x2);
	char x, y, lastdiag, olddiag;
	char s1[MAX_INPUT_LENGTH];
	char s2[MAX_INPUT_LENGTH];
	char column[MAX_INPUT_LENGTH + 1];

	for (int i = 0; i < MAX_INPUT_LENGTH; i++) {
		char c = (x1.entry[i / 6] >> (27 - 5 * (i % 6))) & 0x1F;
		if (c == 0)
			break;
		s1[i] = BEFORE_A_CHAR + c;
	}
	for (int i = 0; i < MAX_INPUT_LENGTH; i++) {
		char c = (x2.entry[i / 6] >> (27 - 5 * (i % 6))) & 0x1F;
		if (c == 0)
			break;
		s2[i] = BEFORE_A_CHAR + c;
	}

	for (y = 1; y <= s1len; y++)
		column[y] = y;
	for (x = 1; x <= s2len; x++) {
		column[0] = x;
		for (y = 1, lastdiag = x - 1; y <= s1len; y++) {
			olddiag = column[y];
			column[y] = MIN3(column[y] + 1, column[y - 1] + 1, lastdiag + (s1[y - 1] == s2[x - 1] ? 0 : 1));
			lastdiag = olddiag;
		}
	}
	return column[s1len];
}

/**
 * calculate Hamming distance of 2 strings in GPU.
 * @param x1 first string
 * @param x2 second string
*/
__device__
char hamming(Int3 x1, Int3 x2) {
	char s1len = (char)len_decode(x1), s2len = (char)len_decode(x2);
	if (s1len != s2len)
		return 77;

	char ans = 0;
	for (int i = 0; i < s1len; i++) {
		char c1 = (x1.entry[i / 6] >> (27 - 5 * (i % 6))) & 0x1F;
		char c2 = (x2.entry[i / 6] >> (27 - 5 * (i % 6))) & 0x1F;
		if (c1 != c2)
			ans++;
	}
	return ans;
}

/**
 * calculate distances of strings from given pairs and flag ones exceeding the threshold.
 *
 * @param seq sequence input
 * @param index pairs of sequence to calculate
 * @param distance Levenshtein/Hamming distance threshold
 * @param measure enum representing Levenshtein/Hamming
 * @param distanceOutput output distance, if null the output won't be written
 * @param flagOutput array output flag
 * @param n array length of index
 * @param seqLen array length of seq
*/
__global__
void cal_distance(Int3* seq, Int2* index, int distance, char measure,
                  char* distanceOutput, char* flagOutput, int n, int seqLen) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid >= n)
		return;

	Int2 indexPair = index[tid];
	if ((distanceOutput != NULL) && (indexPair.x == indexPair.y)) {
		flagOutput[tid] =  0;
		return;
	}

	if (indexPair.x >= seqLen || indexPair.y >= seqLen) {
		printf("curious case! %d %d\n", indexPair.x, indexPair.y);
		flagOutput[tid] =  0;
		return;
	}

	char newOutput = measure == LEVENSHTEIN ?
	                 levenshtein(seq[indexPair.x], seq[indexPair.y]) :
	                 hamming(seq[indexPair.x], seq[indexPair.y]);
	if (distanceOutput != NULL)
		distanceOutput[tid] = newOutput;
	flagOutput[tid] =  newOutput <= distance;
}

/**
 * expand operation part of solving bin packing for 2D buffer.
 *
 * @param matrix statistics of all chunks where each row record the histogram count of each chunk and nRow=nChunk
 * @param output assignment of each chunk to the bins
 * @param nBit bin capacity expressed in log2 form
 * @param nRow number of rows of the matrix
 * @param nColumn number of columns of the matrix
*/
__global__
void gen_assignment(int* matrix, int* output, int nBit, int nRow, int nColumn) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid >= nColumn)
		return;

	size_t ans = 0;
	for (int i = 0; i < nRow; i++)
		ans += matrix[i * nColumn + tid];
	ans = (ans >> nBit);
	if (ans > MAX)
		printf("gen_assignment overflow\n");
	for (int i = 0; i < nRow; i++)
		output[i * nColumn + tid] = ans;
}

/**
 * expand operation part of solving bin packing for lower bound.
 *
 * @param matrix statistics of all chunks where each row record the histogram count of each chunk and nRow=nChunk
 * @param keyOut the regrouping of each bin
 * @param valueOut the upper bound of each grouped bin
 * @param nBit bin capacity expressed in log2 form
 * @param valueMax last sequence index
 * @param nRow number of rows of the matrix
 * @param nColumn number of columns of the matrix
*/
__global__
void gen_bounds(size_t* matrix, int* keyOut, int* valueOut, int nBit, int valueMax, int nRow, int nColumn) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid >= nColumn)
		return;

	size_t intermediate = valueMax;
	valueOut[tid] = intermediate * (tid + 1) / nColumn - 1;

	size_t ans = 0;
	for (int i = 0; i < nRow; i++)
		ans += matrix[i * nColumn + tid];
	ans = (ans >> nBit);
	if (ans > MAX)
		printf("gen_bounds overflow");
	keyOut[tid] = ans;
}

/**
 * flag data to be removed after the lower bound has been processed. This includes both useless group and processed rows.
 *
 * @param valueInput seqIndex input
 * @param valueOffsets group offset
 * @param output flag output
 * @param lowerbound the lowerbound used
 * @param n array length of valueOffsets
*/
__global__
void flag_lowerbound(int* valueInput, int* valueOffsets, char* output, int lowerbound, int n) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid >= n)
		return;

	int start = tid == 0 ? 0 : valueOffsets[tid - 1];
	int end = valueOffsets[tid];
	int validCount = 0;

	for (int i = start; i < end; i++) {
		if (valueInput[i] > lowerbound)
			validCount++;
		else
			output[i] = 0;
	}

	if (validCount < 2)
		for (int i = start; i < end; i++)
			output[i] = 0;
}

/**
 * utility to generate keys for matrix processing.
 *
 * @param output key output with range 0 to n-1 each repeating nRepeat time
 * @param n number of rows
 * @param nRepeat number of columns
*/
__global__
void make_row_index(int* output, int n, int nRepeat) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid >= n)
		return;

	for (int i = tid * nRepeat; i < tid * nRepeat + nRepeat; i++)
		output[i] = tid;
}

/**
 * utility to cast types.
 *
 * @param input input array
 * @param output output array
 * @param n number of rows
*/
__global__
void toSizeT(int* input, size_t* output, int n) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid >= n)
		return;
	output[tid] = input[tid];
}

/**
 * perform binary search with round-down return of index value when not found.
 *
 * @param query value to search
 * @param db database for searching
 * @param dbLen number of rows in db
*/
#ifdef TEST_ENV
__host__
#endif
__device__
int binarySearch(int query, int* db , int dbLen) {
	int start = 0, end = dbLen;
	while ((end - start) > 1) {
		int currentIndex = (end - start) / 2;
		int current = db[currentIndex];
		if (current == query)
			return currentIndex + 1;
		else if (current > query)
			end = currentIndex;
		else
			start = currentIndex + 1;
	}
	return db[start] > query ? start : end;
}

/**
 * turning pairs and frequencies from sequence format to repertoire format.
 *
 * @param pairs pair result from nearest neighbor search
 * @param values returning frequency of the corresponding pair
 * @param seqFreq frequency of each CDR3 sequence
 * @param repSizes size of each repertoire
 * @param repCount number of repertoires
 * @param n number of pairs
*/
__global__
void pair2rep(Int2* pairs, size_t* values, int* seqFreq,
              int* repSizes, int repCount, int n) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid >= n)
		return;
	Int2 pair = pairs[tid];
	int newX = binarySearch(pair.x, repSizes, repCount);
	int newY = binarySearch(pair.y, repSizes, repCount);
	pairs[tid] = {.x = newX, .y = newY};
	if (newX == newY)
		values[tid] = ((size_t)seqFreq[pair.x]) * seqFreq[pair.y] * 2; /*our method only*/
	else
		values[tid] = ((size_t)seqFreq[pair.x]) * seqFreq[pair.y];
}

__global__
void init_overlap_output(Int2* pairOut, size_t* freqOut, int* seqFreq,
                         int* repSizes, int repCount, int n) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid >= n)
		return;

	int rep = binarySearch(tid, repSizes, repCount);
	pairOut[tid] = {.x = rep, .y = rep};
	freqOut[tid] = ((size_t)seqFreq[tid]) * seqFreq[tid];
}