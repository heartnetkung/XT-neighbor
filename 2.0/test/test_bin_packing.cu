#include <stdio.h>
#include "test_util.cu"
#include "../src/xtn_inner.cu"

TEST(bin_packing, {
	int len = 3, nLevel = 3;

	int* histogramInput;
	hipHostMalloc(&histogramInput, sizeof(int) * len * nLevel);
	histogramInput[0] = 1; histogramInput[1] = 2; histogramInput[2] = 3;
	histogramInput[3] = 2; histogramInput[4] = 3; histogramInput[5] = 4;
	histogramInput[6] = 4; histogramInput[7] = 1; histogramInput[8] = 1;
	int* histogramInput_d = host_to_device(histogramInput, len * nLevel);

	int* deviceInt;
	hipMalloc(&deviceInt, sizeof(int));
	MemoryContext ctx;
	ctx.maxThroughputExponent = 4;

	int** output;
	int offsetLen =  solve_bin_packing(histogramInput_d, output, len, nLevel, deviceInt, ctx);

	int expectedOffsetLen = 2;
	int expectedOut[][2] = {{3, 6}, {5, 9}, {5, 6}};

	check(offsetLen == expectedOffsetLen);
	for (int i = 0; i < len; i++)
		for (int j = 0; j < offsetLen; j++)
			check(expectedOut[i][j] == output[i][j]);
})

TEST(cal_lowerbounds, {
	int len = 3, nLevel = 3, seqLen = 35;

	int* histogramInput;
	hipHostMalloc(&histogramInput, sizeof(int) * len * nLevel);
	histogramInput[0] = 1; histogramInput[1] = 2; histogramInput[2] = 3;
	histogramInput[3] = 2; histogramInput[4] = 3; histogramInput[5] = 4;
	histogramInput[6] = 4; histogramInput[7] = 1; histogramInput[8] = 1;
	int* histogramInput_d = host_to_device(histogramInput, len * nLevel);

	int* deviceInt;
	hipMalloc(&deviceInt, sizeof(int));
	MemoryContext ctx;
	ctx.maxThroughputExponent = 4;

	int* output;
	int offsetLen =  cal_lowerbounds(histogramInput_d, output, len, nLevel, seqLen, deviceInt, ctx);

	for (int i = 0; i < offsetLen; i++)
		printf("%d ", output[i]);
	printf("offsetLen: %d", offsetLen);


	// int expectedOffsetLen = 2;
	// int expectedOut[][2] = {{3, 6}, {5, 9}, {5, 6}};

	// check(offsetLen == expectedOffsetLen);
	// for (int i = 0; i < len; i++)
	// 	for (int j = 0; j < offsetLen; j++)
	// 		check(expectedOut[i][j] == output[i][j]);
})
