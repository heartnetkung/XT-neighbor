#include <stdio.h>
#include "test_util.cu"
#include "../src/xtn_inner.cu"

TEST(bin_packing, {
	int len = 3, nLevel = 3;

	int* histogramInput;
	hipHostMalloc((void**) &histogramInput, sizeof(int) * len * nLevel);
	histogramInput[0] = 1; histogramInput[1] = 2; histogramInput[2] = 3;
	histogramInput[3] = 2; histogramInput[4] = 3; histogramInput[5] = 4;
	histogramInput[6] = 4; histogramInput[7] = 1; histogramInput[8] = 1;
	int* histogramInput_d = host_to_device(histogramInput, len * nLevel);

	int* deviceInt;
	hipMalloc((void**)&deviceInt, sizeof(int));

	int expectedLen = 6;
	int expectedOut[] = {3, 6, 5, 9, 5, 6};

	int* output;
	int outputLen =  solve_bin_packing(histogramInput_d, output, 4, len, nLevel, deviceInt);
	int* outputHost = device_to_host(output, outputLen);

	check(outputLen == 6);
	for (int i = 0; i < expectedLen; i++)
		check(expectedOut[i] == outputHost[i]);
})
