#include <stdio.h>
#include "test_util.cu"
#include "../src/xtn_inner.cu"

TEST(bin_packing, {
	int len = 3, n_level = 3;

	int* histogramInput;
	hipHostMalloc(&histogramInput, sizeof(int) * len * n_level);
	histogramInput[0] = 1; histogramInput[1] = 2; histogramInput[2] = 3;
	histogramInput[3] = 2; histogramInput[4] = 3; histogramInput[5] = 4;
	histogramInput[6] = 4; histogramInput[7] = 1; histogramInput[8] = 1;
	int* histogramInput_d = host_to_device(histogramInput, len * n_level);

	int* deviceInt;
	hipMalloc((void**)&deviceInt, sizeof(int));

	size_t* output;
	size_t outputLen =  solve_bin_packing(histogramInput, output, 4, len, n_level, int* deviceInt)

	// printf("outputLen: %lu\n", outputLen);
	// print_size_t_arr(output);
})
