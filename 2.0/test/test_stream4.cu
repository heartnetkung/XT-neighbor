#include "test_util.cu"
#include "../src/xtn_inner.cu"

TEST(Stream4, {
	int seqLen = 5;
	char seqs[seqLen][6] = {"CAAA", "CADA", "CAAA", "CDKD", "CAAK"};
	int pairLen = 10;
	int distance = 1;

	//allocate inputs
	Int3 * seq1d, *seq1h;
	Int2 * pairs_d, *pairs_h;
	XTNOutput output;
	int* deviceInt;
	hipMalloc(&deviceInt, sizeof(int));
	hipMalloc(&seq1d, sizeof(Int3)*seqLen);
	hipHostMalloc(&seq1h, sizeof(Int3)*seqLen);
	hipMalloc(&pairs_d, sizeof(Int2)*pairLen);
	hipHostMalloc(&pairs_h, sizeof(Int2)*pairLen);

	//make inputs
	for (int i = 0; i < seqLen; i++)
		seq1h[i] = str_encode(seqs[i]);
	int count = 0;
	for (int i = 0; i < 5; i++)
		for (int j = i + 1; j < 5; j++)
			pairs_h[count++] = {.x = i, .y = j};
	seq1d = host_to_device(seq1h, seqLen);
	pairs_d = host_to_device(pairs_h, pairLen);

	//do testing
	Chunk<Int2> pairInput;
	pairInput.ptr = pairs_d;
	pairInput.len = pairLen;
	stream_handler4_nn(pairInput, output, seq1d, seqLen, distance, LEVENSHTEIN, deviceInt);

	//expactation
	int expectedLen = 5;
	Int2 expectedPairs[] = {
		{.x = 0, .y = 1}, {.x = 0, .y = 2}, {.x = 0, .y = 4}, {.x = 1, .y = 2}, {.x = 2, .y = 4}
	};
	char expectedDistances[] = {1, 0, 1, 1, 1};

	//check
	check(output.len == expectedLen);
	check_arr(output.indexPairs, expectedPairs, output.len);
	check_arr(output.pairwiseDistances, expectedDistances, output.len);
})