#include "test_util.cu"
#include "../src/xtn_inner.cu"

TEST(Stream4, {
	int seqLen = 5;
	char seqs[seqLen][6] = {"CAAA", "CADA", "CAAA", "CDKD", "CAAK"};
	int pairLen = 10;
	int distance = 1;

	//allocate inputs
	Int3 * seq1d, *seq1h;
	Int2 * pairs_d, *pairs_h;
	XTNOutput output;
	int* deviceInt;
	hipMalloc(&deviceInt, sizeof(int));
	hipMalloc(&seq1d, sizeof(Int3)*seqLen);
	hipHostMalloc(&seq1h, sizeof(Int3)*seqLen);
	hipMalloc(&pairs_d, sizeof(Int2)*pairLen);
	hipHostMalloc(&pairs_h, sizeof(Int2)*pairLen);

	//make inputs
	for (int i = 0; i < seqLen; i++)
		seq1h[i] = str_encode(seqs[i]);
	int count = 0;
	for (int i = 0; i < 5; i++)
		for (int j = i + 1; j < 5; j++)
			pairs_h[count++] = {.x = i, .y = j};
	seq1d = host_to_device(seq1h, seqLen);
	pairs_d = host_to_device(pairs_h, pairLen);

	//do testing
	Chunk<Int2> pairInput;
	pairInput.ptr = pairs_d;
	pairInput.len = pairLen;
	stream_handler4_nn(pairInput, output, seq1d, seqLen, distance, LEVENSHTEIN, deviceInt);

	//expactation
	int expectedLen = 5;
	Int2 expectedPairs[] = {
		{.x = 0, .y = 1}, {.x = 0, .y = 2}, {.x = 0, .y = 4}, {.x = 1, .y = 2}, {.x = 2, .y = 4}
	};
	char expectedDistances[] = {1, 0, 1, 1, 1};

	//check
	check(output.len == expectedLen);
	for (int i = 0; i < expectedLen; i++) {
		check(expectedPairs[i].x == output.indexPairs[i].x);
		check(expectedPairs[i].y == output.indexPairs[i].y);
		check(expectedDistances[i] == output.pairwiseDistances[i]);
	}
})