#include "test_util.cu"
#include "../src/xtn_inner.cu"

TEST(Stream4Overlap, {
	int seqLen = 4;
	int infoLen = 5;
	// original sequence char seqs[seqLen][6] = {"CAAA", "CADA", "CAAA", "CDKD", "CAAK"};
	char seqs[seqLen][6] = {"CAAA", "CADA", "CDKD", "CAAK"};
	SeqInfo info_h[] = {
		{.frequency = 3, .repertoire = 0}, {.frequency = 5, .repertoire = 1},
		{.frequency = 4, .repertoire = 0}, {.frequency = 6, .repertoire = 1}, {.frequency = 7, .repertoire = 1}
	};
	int inputOffsets[] = {2, 3, 4, 5};
	int pairLen = 6;
	int distance = 1;

	//allocate inputs
	Int3 * seq_d, *seq_h;
	Int2 * pairs_d, *pairs_h;
	std::vector<XTNOutput> allOutputs;
	int* deviceInt, *inputOffsets_d;
	SeqInfo* info_d;
	MemoryContext ctx;
	hipMalloc(&deviceInt, sizeof(int));
	hipMalloc(&seq_d, sizeof(Int3)*seqLen);
	hipHostMalloc(&seq_h, sizeof(Int3)*seqLen);
	hipMalloc(&pairs_d, sizeof(Int2)*pairLen);
	hipHostMalloc(&pairs_h, sizeof(Int2)*pairLen);
	hipMalloc(&info_d, sizeof(SeqInfo)*infoLen);
	ctx.bandwidth2 = 100;

	//make inputs
	for (int i = 0; i < seqLen; i++)
		seq_h[i] = str_encode(seqs[i]);
	int count = 0;
	for (int i = 0; i < seqLen; i++)
		for (int j = i + 1; j < seqLen; j++)
			pairs_h[count++] = {.x = i, .y = j};
	seq_d = host_to_device(seq_h, seqLen);
	pairs_d = host_to_device(pairs_h, pairLen);
	info_d = host_to_device(info_h, infoLen);
	inputOffsets_d = host_to_device(inputOffsets, seqLen);

	// do testing
	Chunk<Int2> pairInput;
	pairInput.ptr = pairs_d;
	pairInput.len = pairLen;
	stream_handler4_overlap(pairInput, allOutputs, seq_d, info_d, inputOffsets_d,
	                        seqLen, distance, LEVENSHTEIN, deviceInt, ctx);
	XTNOutput output = allOutputs.back();

	// checking
	int expectedLen = 3, expectedCount = 1;
	check(output.len == expectedLen);
	check(allOutputs.size() == expectedCount);

	Int2 expectedIndexPair[] = {{.x = 0, .y = 0}, {.x = 0, .y = 1}, {.x = 1, .y = 1}};
	check_device_arr(output.indexPairs, expectedIndexPair, output.len);

	size_t expectedFrequency[] = {24, 41, 70};
	check_device_arr(output.pairwiseFrequencies, expectedFrequency, output.len);
})