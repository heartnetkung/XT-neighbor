#include "test_util.cu"
#include "../src/xtn_inner.cu"

TEST(Stream1, {
	int seqLen = 4;
	char seqs[seqLen][6] = {"CAAA", "CADA", "CAAA", "CDKD"};
	int distance = 1;
	printf("1\n");

	//allocate inputs
	Int3 * seq1d, *seq1h;
	int* histogramOutput;
	hipMalloc((void**)&seq1d, sizeof(Int3)*seqLen);
	hipHostMalloc((void**)&seq1h, sizeof(Int3)*seqLen);
	printf("2\n");

	//make inputs
	for (int i = 0; i < seqLen; i++)
		seq1h[i] = str_encode(seqs[i]);
	seq1d = host_to_device(seq1h, seqLen);

	//do testing
	Chunk<Int3> input;
	input.ptr = seq1d;
	input.len = seqLen;
	int* indexOutput;
	Int3* deletionsOutput;
	int outputLen;
	stream_handler1(input, deletionsOutput, indexOutput, histogramOutput, outputLen, distance);
	printf("3\n");

	//expactation
	int expectedLen = 20;
	char expectedPairs[][5] = {
		"AAA", "CAA", "CAA", "CAA", "CAAA",
		"ADA", "CDA", "CAA", "CAD", "CADA",
		"AAA", "CAA", "CAA", "CAA", "CAAA",
		"DKD", "CKD", "CDD", "CDK", "CDKD",
	};
	int expectedIndex[] = {0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 2, 2, 2, 2, 2, 3, 3, 3, 3, 3};

	//check
	check(outputLen == expectedLen);
	for (int i = 0; i < expectedLen; i++) {
		printf("4\n");
		checkstr(expectedPairs[i], str_decode(deletionsOutput[i]));
		printf("5\n");
		check(expectedIndex[i] == indexOutput[i]);
	}
})