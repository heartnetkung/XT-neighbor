#include "test_util.cu"
#include "../src/xtn_inner.cu"

TEST(OverlapInit, {
	XTNOutput output;
	int* buffer, *infoOffset;
	Int3 * seq1d, *seq1h, *seqOut;
	SeqInfo* infoD;

	int seqLen = 4;
	char seqs[seqLen][6] = {"CAAA", "CADA", "CAAA", "CDKD"};
	SeqInfo info[] = {
		{.frequency = 3, .repertoire = 0}, {.frequency = 4, .repertoire = 0},
		{.frequency = 5, .repertoire = 1}, {.frequency = 6, .repertoire = 1}
	};

	hipMalloc(&buffer, sizeof(int));
	hipMalloc(&seq1d, sizeof(Int3)*seqLen);
	hipHostMalloc(&seq1h, sizeof(Int3)*seqLen);
	hipMalloc(&infoD, sizeof(SeqInfo)*seqLen);

	//make inputs
	for (int i = 0; i < seqLen; i++)
		seq1h[i] = str_encode(seqs[i]);
	seq1d = host_to_device(seq1h, seqLen);
	infoD = host_to_device(info, seqLen);


	// int output = overlap_mode_init(seq1d, seqOut, infoD, &infoOffset,
	//                                output, seqLen, buffer);
})