#include "hip/hip_runtime.h"
#include <stdio.h>
#include "xtn.cu"
#include "brute_force.cu"
#include <locale.h>

const char VERSION[] = "1.0.0\n";
const char HELP_TEXT[] = "xt_neighbor\n"
                         "\t description: perform xt_neighbor algorithm for near neighbor search of T cell receptor's CDR3 sequences\n"
                         "\t -p or --input-path [str] (required): set the path of input file which is a text file containing one CDR3 sequence per line\n"
                         "\t -n or --input-length [number] (required): set the number of sequences given in the input file\n"
                         "\t -d or --distance [number]: set the distance threshold defining the neighbor\n"
                         "\t -c or --check-output: check if the output is correct using brute force algorithm (very slow!)\n"
                         "\t -o or --output-path [str]: set the path of the output file (default to no output)\n"
                         "\t -v or --version: print the version of the program then exit\n"
                         "\t -h or --help: print the help text of the program then exit\n"
                         "\t -V or --verbose: print extra detail as the program runs for debugging purpose\n";

int parse_args(int argc, char **argv, XTNArgs* ans) {
	char* current;

	for (int i = 1; i < argc; i++) {
		current = argv[i];

		if (strcmp(current, "-v") == 0 || strcmp(current, "--version") == 0) {
			printf("%s", VERSION);
			return EXIT;
		}
		else if (strcmp(current, "-h") == 0 || strcmp(current, "--help") == 0) {
			printf("%s", HELP_TEXT);
			return EXIT;
		}
		else if (strcmp(current, "-V") == 0 || strcmp(current, "--verbose") == 0)
			ans->verbose = 1;
		else if (strcmp(current, "-c") == 0 || strcmp(current, "--check-output") == 0)
			ans->checkOutput = 1;
		else if (strcmp(current, "-p") == 0 || strcmp(current, "--input-path") == 0)
			ans->seq1Path = argv[++i];
		else if (strcmp(current, "-o") == 0 || strcmp(current, "--output-path") == 0)
			ans->outputPath = argv[++i];
		else if (strcmp(current, "-d") == 0 || strcmp(current, "--distance") == 0) {
			int distance = ans->distance = atoi(argv[++i]);
			if (distance < 1 || distance > MAX_DISTANCE)
				return print_err("distance must be a valid number ranging from 1-2");
		}
		else if (strcmp(current, "-n") == 0 || strcmp(current, "--input-length") == 0) {
			ans->seq1Len = atoi(argv[++i]);
			if (ans->seq1Len == 0)
				return print_err("invalid input length");
		}
		else
			return print_err("unknown option");
	}

	if (ans->seq1Path == NULL)
		return print_err("missing path for seq1");
	if (ans->seq1Len == 0)
		return print_err("missing length for seq1");

	return SUCCESS;
}

/**
 * read and parse text file to Int3*
*/
int parse_file(char* path, Int3* result, int len) {
	FILE* file = fopen(path, "r");
	if (file == NULL)
		return print_err("file reading failed");

	const int BUFFER_SIZE = 50;
	char line[BUFFER_SIZE];
	int lineNumber = 0, inputCount = 0;
	Int3 newInt3;

	while (fgets(line, BUFFER_SIZE, file)) {
		lineNumber++;
		if (strcmp(line, "\n") == 0 || strcmp(line, " \n") == 0)
			continue;

		newInt3 = str_encode(line);
		if (newInt3.entry[0] == 0) {
			fclose(file);
			char msg[100];
			sprintf(msg, "parsing error at line: %d (only upper-cased amino acids with max length of %d are allowed)", lineNumber, MAX_INPUT_LENGTH);
			return print_err(msg);
		}

		result[inputCount++] = newInt3;
	}

	if (inputCount != len)
		return print_err("input length doesn't match with the actual");

	fclose(file);
	return SUCCESS;
}

/**
 * write XTNOutput to file
*/
int write_file(char* path, XTNOutput output) {
	FILE* file = fopen(path, "w");
	if (file == NULL)
		return print_err("file reading failed");

	Int2 current;
	for (int i = 0; i < output.len; i++) {
		current = output.indexPairs[i];
		fprintf(file, "%d %d %d\n", current.x, current.y , output.pairwiseDistances[i]);
	}

	fclose(file);
	return SUCCESS;
}

int main(int argc, char **argv) {
	int returnCode;
	XTNArgs args;
	Int3* seq1;
	XTNOutput output;

	// 1. parse command line arguments
	returnCode = parse_args(argc, argv, &args);
	if (returnCode != SUCCESS)
		return returnCode;

	// 2. read input
	hipHostMalloc((void**)&seq1, sizeof(Int3) * args.seq1Len);
	returnCode = parse_file(args.seq1Path, seq1, args.seq1Len);
	if (returnCode != SUCCESS) {
		hipFree(seq1);
		return returnCode;
	}
	if (args.verbose)
		print_args(args);

	// 3. perform algorithm
	setlocale(LC_ALL, "");
	xtn_perform(args, seq1, &output);

	// 4. write output, if requested
	if (args.outputPath != NULL) {
		returnCode = write_file(args.outputPath, output);
		if (returnCode != SUCCESS)
			print_err("file writing failed");
	}

	// 5. check output, if requested
	if (args.checkOutput) {
		auto answer = pairwise_distance(seq1, args.seq1Len, args.distance);
		int success = check_intput(answer, output);
		if (success)
			printf("output is verrified to be correct\n");
		else
			print_err("input checking failed");
	}

	// 6. clean up
	hipHostFree(seq1);
	xtn_free(&output);
	printf("Success! Number of triplet: %'zu\n", output.len);
	return 0;
}