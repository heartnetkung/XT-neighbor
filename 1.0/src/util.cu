#include "xtn.h"

int print_err(const char* str) {
#ifndef TEST_ENV
	fprintf(stderr, "Error: %s\n", str);
#endif
	return ERROR;
}

void print_int3(Int3* seqs, int len, char prefix) {
	int n_elements = len < 5 ? len : 5;
	for (int i = 0; i < n_elements; i++) {
		unsigned int* entry = seqs[i].entry;
		printf("%c %08X %08X %08X \n", prefix, entry[0], entry[1], entry[2]);
	}
}

void print_args(XTNArgs args) {
	printf("XTNArgs{\n");
	printf("\tdistance: %d\n", args.distance);
	printf("\tverbose: %d\n", args.verbose);
	printf("\tseq1Len: %d\n", args.seq1Len);
	printf("\tseq1Path: \"%s\"\n", args.seq1Path);
	printf("\toutputPath: \"%s\"\n", args.outputPath);
	printf("\tcheckOutput: %d\n", args.checkOutput);
	printf("}\n");
}

void print_int_arr(int* arr, int n) {
	for (int i = 0; i < n; i++)
		printf("%d ", arr[i]);
	printf("\n");
}

void print_char_arr(char* arr, int n) {
	for (int i = 0; i < n; i++)
		printf("%d ", arr[i]);
	printf("\n");
}

void print_int2_arr(Int2* arr, int n) {
	for (int i = 0; i < n; i++)
		printf("(%d,%d) ", arr[i].x, arr[i].y);
	printf("\n");
}

void _cudaFree(void* a) {
	hipFree(a);
}
void _cudaFree(void* a, void* b) {
	hipFree(a);
	hipFree(b);
}
void _cudaFree(void* a, void* b, void* c) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
}
void _cudaFree(void* a, void* b, void* c, void* d) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
	hipFree(e);
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e, void* f) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
	hipFree(e);
	hipFree(f);
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e, void* f, void* g) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
	hipFree(e);
	hipFree(g);
}

void print_tp(int verbose, const char* step, int throughput) {
	if (verbose)
		printf("step %s completed with throughput: %'d\n", step, throughput);
}

void _cudaFreeHost(void* a, void* b) {
	hipHostFree(a);
	hipHostFree(b);
}

void _free(void* a, void* b, void* c) {
	free(a);
	free(b);
	free(c);
}

int divideCeil(int a, int b) {
	return (a + b - 1) / b;
}

template <typename T>
T* device_to_host(T* arr, int n) {
	T* temp;
	int tempBytes = sizeof(T) * n;
	hipHostMalloc(&temp, tempBytes);
	hipMemcpy(temp, arr, tempBytes, hipMemcpyDeviceToHost);
	return temp;
}

template <typename T>
T* host_to_device(T* arr, int n) {
	T* temp;
	int tempBytes = sizeof(T) * n;
	hipMalloc(&temp, tempBytes);
	hipMemcpy(temp, arr, tempBytes, hipMemcpyHostToDevice);
	return temp;
}

#define gpuerr() { print_cuda_error( __FILE__, __LINE__); }

void print_cuda_error(const char *file, int line) {
	hipError_t code = hipGetLastError();
	if (code != hipSuccess)
		printf("Cuda error at %s %s %d\n", hipGetErrorName(code), file, line);
}

